#include "hip/hip_runtime.h"
#pragma once
#include"cudaLibs.h"
__device__ struct bivec4 {
	float xy;
	float xz;
	float xw;
	float yz;
	float yw;
	float zw;

	__device__ explicit bivec4(float xy = 0.f, float xz = 0.f, float xw = 0.f, float yz = 0.f, float yw = 0.f, float zw = 0.f)
	{
		this->xy = xy, this->xz = xz, this->xw = xw, this->yz = yz, this->yw = yw, this->zw = zw;
	}

	__device__ bivec4 zero()
	{
		return bivec4(0.f, 0.f, 0.f, 0.f, 0.f, 0.f);
	}

	__device__ bivec4 reverse()
	{
		return bivec4(-xy, -xz, -xw, -yz, -yw, -zw);
	}

	//bivec4 radians()
	//{
	//	return bivec4(glm::radians(this->xy), glm::radians(this->xz), glm::radians(this->xw), glm::radians(this->yz), glm::radians(this->yw), glm::radians(this->zw));
	//}

	__device__ 	bivec4 operator+(const bivec4 q)
	{
		return bivec4(xy + q.xy, xz + q.xz, xw + q.xw, yz + q.yz, yw + q.yw, zw + q.zw);
	}

	__device__ bivec4 operator+=(const bivec4 q)
	{
		return bivec4(xy + q.xy, xz + q.xz, xw + q.xw, yz + q.yz, yw + q.yw, zw + q.zw);
	}

	__device__ bivec4 operator-(const bivec4 q)
	{
		return bivec4(xy - q.xy, xz - q.xz, xw - q.xw, yz - q.yz, yw - q.yw, zw - q.zw);
	}

	__device__ 	bivec4 operator-=(const bivec4 q)
	{
		return bivec4(xy - q.xy, xz - q.xz, xw - q.xw, yz - q.yz, yw - q.yw, zw - q.zw);
	}

	__device__ bivec4 operator*(const bivec4 q)
	{
		return bivec4(xy * q.xy, xz * q.xz, xw * q.xw, yz * q.yz, yw * q.yw, zw * q.zw);
	}

	__device__ bivec4 operator*(const float f)
	{
		return bivec4(xy * f, xz * f, xw * f, yz * f, yw * f, zw * f);
	}

	__device__ bivec4 operator*=(const bivec4 q)
	{
		return bivec4(xy * q.xy, xz * q.xz, xw * q.xw, yz * q.yz, yw * q.yw, zw * q.zw);
	}

	__device__ bivec4 operator*=(const float f)
	{
		return bivec4(xy * f, xz * f, xw * f, yz * f, yw * f, zw * f);
	}

	__device__ bivec4 operator/(const bivec4 q)
	{
		return bivec4(xy / q.xy, xz / q.xz, xw / q.xw, yz / q.yz, yw / q.yw, zw / q.zw);
	}

	__device__ bivec4 operator/(const float f)
	{
		return bivec4(xy / f, xz / f, xw / f, yz / f, yw / f, zw / f);
	}

	__device__ bivec4 operator/=(const bivec4 q)
	{
		return bivec4(xy / q.xy, xz / q.xz, xw / q.xw, yz / q.yz, yw / q.yw, zw / q.zw);
	}

	__device__ bivec4 operator/=(const float f)
	{
		return bivec4(xy / f, xz / f, xw / f, yz / f, yw / f, zw / f);
	}
};

__device__ struct quadvec4 {
	float xyzw;

	__device__ explicit quadvec4(float xyzw = 0.f)
	{
		this->xyzw = xyzw;
	}

	__device__ quadvec4 zero()
	{
		return quadvec4(0.f);
	}

	__device__ quadvec4 one()
	{
		return quadvec4(1.f);
	}

	__device__ bivec4 mul_bv(bivec4 b)
	{
		return bivec4(-b.zw * xyzw, b.yw * xyzw, -b.yz * xyzw, -b.xw * xyzw, b.xz * xyzw, -b.xy * xyzw);
	}

	__device__ quadvec4 operator+(const quadvec4 q)
	{
		return quadvec4(xyzw + q.xyzw);
	}
	__device__ quadvec4 operator+=(const quadvec4 q)
	{
		return quadvec4(xyzw + q.xyzw);
	}

	__device__ quadvec4 operator*(const quadvec4 q)
	{
		return quadvec4(xyzw * q.xyzw);
	}

	__device__ quadvec4 operator*=(const quadvec4 q)
	{
		return quadvec4(xyzw * q.xyzw);
	}

	__device__ quadvec4 operator*(const float f)
	{
		return quadvec4(xyzw * f);
	}

	__device__ quadvec4 operator*=(const float f)
	{
		return quadvec4(xyzw * f);
	}

};

__device__ struct rotor4
{
	float s;
	bivec4 b;
	quadvec4 q;

	__device__ explicit rotor4(float s = 1.f, bivec4 b = bivec4(0.f, 0.f, 0.f, 0.f, 0.f, 0.f), quadvec4 q = quadvec4(0.f))
	{
		this->s = s;
		this->b = b;
		this->q = q;
	}

	__device__ void set(float s = 1.f, bivec4 b = bivec4(0.f, 0.f, 0.f, 0.f, 0.f, 0.f), quadvec4 q = quadvec4(0.f))
	{
		this->s = s;
		this->b = b;
		this->q = q;
	}

	__device__ rotor4 reverse()
	{
		return rotor4(s, b.reverse(), q);
	}

	__device__ float mul_bv_s(bivec4 c)
	{
		return
			-b.xy * c.xy
			- b.xz * c.xz
			- b.xw * c.xw
			- b.yz * c.yz
			- b.yw * c.yw
			- b.zw * c.zw;
	}

	__device__ bivec4 mul_bv_b(bivec4 c)
	{
		return bivec4(
			-b.xw * c.yw - b.xz * c.yz + b.yw * c.xw + b.yz * c.xz,
			-b.xw * c.zw + b.xy * c.yz - b.yz * c.xy + b.zw * c.xw,
			b.xy * c.yw + b.xz * c.zw - b.yw * c.xy - b.zw * c.xz,
			-b.xy * c.xz + b.xz * c.xy - b.yw * c.zw + b.zw * c.yw,
			b.xw * c.xy - b.xy * c.xw + b.yz * c.zw - b.zw * c.yz,
			b.xw * c.xz - b.xz * c.xw + b.yw * c.yz - b.yz * c.yw
		);
	}

	__device__ quadvec4 mul_bv_q(bivec4 c)
	{
		return quadvec4(
			b.xw * c.yz
			+ b.xy * c.zw
			- b.xz * c.yw
			- b.yw * c.xz
			+ b.yz * c.xw
			+ b.zw * c.xy);
	}

	__device__ rotor4 mul_bv(bivec4 c)
	{
		return rotor4(mul_bv_s(c), mul_bv_b(c), mul_bv_q(c));
	}

	__device__ rotor4 operator+(const rotor4 r)
	{
		return rotor4(s + r.s, b + r.b, q + r.q);
	}
	__device__ rotor4 operator+=(const rotor4 r)
	{
		return rotor4(s + r.s, b + r.b, q + r.q);
	}
	__device__ rotor4 operator*(const rotor4 r1)
	{
		bivec4 rb(r1.b);
		quadvec4 rq(r1.q);
		float a0(mul_bv_s(r1.b));
		bivec4 a2(mul_bv_b(r1.b));
		quadvec4 a4(mul_bv_q(r1.b));
		return rotor4(s * r1.s + a0 + q.xyzw * r1.q.xyzw,
			rb * s
			+ b * r1.s
			+ a2
			+ q.mul_bv(rb)
			+ rq.mul_bv(b),
			rq * s + q * r1.s + a4);
	}
	__device__ rotor4 operator*=(const rotor4 r1)
	{
		bivec4 rb(r1.b);
		quadvec4 rq(r1.q);
		float a0(mul_bv_s(r1.b));
		bivec4 a2(mul_bv_b(r1.b));
		quadvec4 a4(mul_bv_q(r1.b));
		return rotor4(s * r1.s + a0 + q.xyzw * r1.q.xyzw,
			rb * s
			+ b * r1.s
			+ a2
			+ q.mul_bv(rb)
			+ rq.mul_bv(b),
			rq * s + q * r1.s + a4);
	}
};

__device__ struct trivec4
{
	float xyz;
	float xyw;
	float xzw;
	float yzw;
	__device__ explicit trivec4(float xyz = 0.f, float xyw = 0.f, float xzw = 0.f, float yzw = 0.f)
	{
		this->xyz = xyz;
		this->xyw = xyw;
		this->xzw = xzw;
		this->yzw = yzw;
	}
	__device__ trivec4 operator+(const trivec4 q)
	{
		return trivec4(xyz + q.xyz, xyw + q.xyw, xzw + q.xzw, yzw + q.yzw);
	}
	__device__ trivec4 operator+=(const trivec4 q)
	{
		return trivec4(xyz + q.xyz, xyw + q.xyw, xzw + q.xzw, yzw + q.yzw);
	}
};

__device__ struct vec2
{
	float x;
	float y;
	__device__ explicit vec2(float x = 0.f, float y = 0.f)
	{
		this->x = x;
		this->y = y;
	}

	__device__ vec2 operator+(const vec2 q)
	{
		return vec2(x + q.x, y + q.y);
	}
	__device__ vec2 operator+=(const vec2 q)
	{
		return vec2(x + q.x, y + q.y);
	}
	__device__ vec2 operator-(const vec2 q)
	{
		return vec2(x - q.x, y - q.y);
	}
	__device__ vec2 operator-=(const vec2 q)
	{
		return vec2(x - q.x, y - q.y);
	}
	__device__ vec2 operator*(const vec2 q)
	{
		return vec2(x * q.x, y * q.y);
	}
	__device__ vec2 operator*=(const vec2 q)
	{
		return vec2(x * q.x, y * q.y);
	}
	__device__ vec2 operator*(const float f)
	{
		return vec2(x * f, y * f);
	}
	__device__ vec2 operator*=(const float f)
	{
		return vec2(x * f, y * f);
	}
	__device__ vec2 operator/(const float f)
	{
		return vec2(x / f, y / f);
	}	
	__device__ float operator[ ](int n)
	{
		if (n == 0) return this->x;
		else if (n == 1) return this->y;
	}
};

__device__ struct vec3
{
	float x;
	float y;
	float z;
	__device__ explicit vec3(float x = 0.f, float y = 0.f, float z = 0.f)
	{
		this->x = x;
		this->y = y;
		this->z = z;
	}

	__device__ vec3 operator+(const vec3 q)
	{
		return vec3(x + q.x, y + q.y, z + q.z);
	}
	__device__ vec3 operator+=(const vec3 q)
	{
		return vec3(x + q.x, y + q.y, z + q.z);
	}
	__device__ vec3 operator-(const vec3 q)
	{
		return vec3(x - q.x, y - q.y, z - q.z);
	}
	__device__ vec3 operator-=(const vec3 q)
	{
		return vec3(x - q.x, y - q.y, z - q.z);
	}
	__device__ vec3 operator*(const vec3 q)
	{
		return vec3(x * q.x, y * q.y, z * q.z);
	}
	__device__ vec3 operator*=(const vec3 q)
	{
		return vec3(x * q.x, y * q.y, z * q.z);
	}
	__device__ vec3 operator*(const float f)
	{
		return vec3(x * f, y * f, z * f);
	}
	__device__ vec3 operator*=(const float f)
	{
		return vec3(x * f, y * f, z * f);
	}
	__device__ vec3 operator/(const float f)
	{
		return vec3(x / f, y / f, z / f);
	}
	__device__ float operator[ ](int n)
	{
		if (n == 0) return this->x;
		else if (n == 1) return this->y;
		else if (n == 2) return this->z;
	}
};

__device__ struct vec4
{
	float x;
	float y;
	float z;
	float w;
	__device__ explicit vec4(float x, float y, float z, float w)
	{
		this->x = x;
		this->y = y;
		this->z = z;
		this->w = w;
	}

	__device__ explicit vec4(float n = 0.f)
	{
		this->x = n;
		this->y = n;
		this->z = n;
		this->w = n;
	}
	
	__device__ vec4 operator+(const vec4 q)
	{
		return vec4(x + q.x, y + q.y, z + q.z, w + q.w);
	}
	__device__ vec4 operator+=(const vec4 q)
	{
		return vec4(x + q.x, y + q.y, z + q.z, w + q.w);
	}
	__device__ vec4 operator-(const vec4 q)
	{
		return vec4(x - q.x, y - q.y, z - q.z, w - q.w);
	}
	__device__ vec4 operator-=(const vec4 q)
	{
		return vec4(x - q.x, y - q.y, z - q.z, w - q.w);
	}
	__device__ vec4 operator-()
	{
		return vec4(-x, -y, -z, -w);
	}
	__device__ vec4 operator*(const vec4 q)
	{
		return vec4(x * q.x, y * q.y, z * q.z, w * q.w);
	}
	__device__ vec4 operator*=(const vec4 q)
	{
		return vec4(x * q.x, y * q.y, z * q.z, w * q.w);
	}
	__device__ vec4 operator*(const float f)
	{
		return vec4(x * f, y * f, z * f, w * f);
	}
	__device__ vec4 operator*=(const float f)
	{
		return vec4(x * f, y * f, z * f, w * f);
	}
	__device__ vec4 operator/(const float f)
	{
		return vec4(x / f, y / f, z / f, w / f);
	}

	__device__ vec4 operator/(const vec4 v)
	{
		return vec4(x / v.x, y / v.y, z / v.z, w / v.w);
	}

	__device__ bool operator==(const vec4 q)
	{
		return x == q.x && y == q.y && z == q.z && w == q.w;
	}

	__device__ bool operator!=(const vec4 q)
	{
		return x != q.x || y != q.y || z != q.z || w != q.w;
	}

	__device__ float operator[ ](int n)
	{
		if (n == 0) return this->x;
		else if (n == 1) return this->y;
		else if (n == 2) return this->z;
		else if (n == 3) return this->w;
	}

	__device__ void value(int n, float n2)
	{
		if (n == 0) this->x = n2;
		else if (n == 1) this->y = n2;
		else if (n == 2) this->z = n2;
		else if (n == 3) this->w = n2;
	}
};

__device__ static struct mat4x3
{
	vec3 x;
	vec3 y;
	vec3 z;
	vec3 w;
	//explicit mat4x3(vec4 x, vec4 y, vec4 z)
	//{
	//	this->x = x;
	//	this->y = y;
	//	this->z = z;
	//}
	__device__ vec3 operator[ ](int n)
	{
		if (n == 0) return this->x;
		else if (n == 1) return this->y;
		else if (n == 2) return this->z;
		else if (n == 3) return this->w;
	}
	__device__ void value(int n, vec3 v)
	{
		if (n == 0) this->x = v;
		else if (n == 1) this->y = v;
		else if (n == 2) this->z = v;
		else if (n == 3) this->w = v;
	}
};

__device__ static struct mat4
{
	vec4 x;
	vec4 y;
	vec4 z;
	vec4 w;
	//explicit mat4(vec4 x, vec4 y, vec4 z, vec4 w)
	//{
	//	this->x = x;
	//	this->y = y;
	//	this->z = z;
	//	this->w = w;
	//}
	__device__ vec4 operator[ ](int n)
	{
		if (n == 0) return this->x;
		else if (n == 1) return this->y;
		else if (n == 2) return this->z;
		else if (n == 3) return this->w;
	}
};

__device__ static struct ivec2
{
	int x;
	int y;
	explicit ivec2(int x, int y)
	{
		this->x = x;
		this->y = y;
	}
	__device__ bool operator==(const ivec2 q)
	{
		return x == q.x && y == q.y;
	}
	__device__ int operator[ ](int n)
	{
		if (n == 0) return this->x;
		else if (n == 1) return this->y;
	}
	__device__ void value(int n, int n2)
	{
		if (n == 0) this->x = n2;
		else if (n == 1) this->y = n2;
	}
};

__device__ static struct ivec4
{
	int x;
	int y;
	int z;
	int w;
	//__device__ explicit ivec4(int x, int y, int z, int w)
	//{
	//	this->x = x;
	//	this->y = y;
	//	this->z = z;
	//	this->w = w;
	//}
	__device__ bool operator==(const ivec4 q)
	{
		return x == q.x && y == q.y && z == q.z && w == q.w;
	}
	__device__ int operator[ ](int n)
	{
		if (n == 0) return this->x;
		else if (n == 1) return this->y;
		else if (n == 2) return this->z;
		else if (n == 3) return this->w;
	}
	__device__ void value(int n, int n2)
	{
		if (n == 0) this->x = n2;
		else if (n == 1) this->y = n2;
		else if (n == 2) this->z = n2;
		else if (n == 3) this->w = n2;
	}
};

__device__ static float _d_max(float x, float y)
{
	return x > y ? x : y;
}

__device__ static int _d_max(int x, int y)
{
	return x > y ? x : y;
}

__device__ static vec3 _d_max(vec3 x, vec3 y)
{
	return vec3(_d_max(x.x, y.x), _d_max(x.y, y.y), _d_max(x.z, y.z));
}

__device__ static float _d_min(float x, float y)
{
	return x < y ? x : y;
}

__device__ static int _d_min(int x, int y)
{
	return x < y ? x : y;
}

__device__ static float _d_abs(float x)
{
	return x > 0.f ? x : -x;
}

__device__ static vec4 abs(vec4 x)
{
	return vec4(_d_abs(x.x), _d_abs(x.y), _d_abs(x.z), _d_abs(x.w));
}

__device__ static int _d_abs(int x)
{
	return x > 0 ? x : -x;
}

__device__ static float _d_fmod(float x, float y)
{
	return x - int(x / y) * y;
}

__device__ static float _d_fract(float x)
{
	return x - int(x);
}

__device__ static float _d_sign(float x)
{
	return x > 0.f ? 1.f : -1.f;
}

__device__ static int _d_sign(int x)
{
	return x > 0 ? 1 : -1;
}

__device__ static vec4 sign(vec4 x)
{
	return vec4(_d_sign(x.x), _d_sign(x.y), _d_sign(x.z), _d_sign(x.w));
}

__device__ static float _d_clamp(float x, float y, float z)
{
	return x < y ? y : x > z ? z : x;
}

__device__ static int _d_clamp(int x, int y, int z)
{
	return x < y ? y : x > z ? z : x;
}

__device__ static int _d_inverse_clamp(int x, int y, int z)
{
	return x < y ? z : x > z ? y : x;
}

__device__ static float length(vec2 x)
{
	return sqrt(x.x * x.x + x.y * x.y);
}

__device__ static float length(vec3 x)
{
	return sqrt(x.x * x.x + x.y * x.y + x.z * x.z);
}

__device__ static float length(vec4 x)
{
	return sqrt(x.x * x.x + x.y * x.y + x.z * x.z + x.w * x.w);
}

__device__ static vec2 normalize(vec2 x)
{
	return x / length(x);
}

__device__ static vec3 normalize(vec3 x)
{
	return x / length(x);
}

__device__ static vec4 normalize(vec4 x)
{
	return x / length(x);
}

__device__ static float dot(vec3 x, vec3 y)
{
	return x.x * y.x + x.y * y.y + x.z * y.z;
}

__device__ static float dot(vec4 x, vec4 y)
{
	return x.x * y.x + x.y * y.y + x.z * y.z + x.w * y.w;
}

__device__ static float _d_mix(float x, float y, float n)
{
	return x * (1.f - n) + y * n;
}

__device__ static vec4 mix(vec4 x, vec4 y,float n)
{
	return vec4(_d_mix(x.x, y.x, n), _d_mix(x.y, y.y, n), _d_mix(x.z, y.z, n), _d_mix(x.w, y.w, n));
}

__device__ static vec4 left_contract_bv(vec4 v, bivec4 b) {
	return
		vec4(
			-v.y * b.xy - v.z * b.xz - v.w * b.xw,
			v.x * b.xy - v.z * b.yz - v.w * b.yw,
			v.x * b.xz + v.y * b.yz - v.w * b.zw,
			v.x * b.xw + v.y * b.yw + v.z * b.zw
		);
}

//__device__ static vec4 left_contract_bv(vec4 v, float* bs, int index) {
//	return
//		vec4(
//			-v.y * bs[6 * index] - v.z * bs[6 * index + 1] - v.w * bs[6 * index + 2],
//			v.x * bs[6 * index] - v.z * bs[6 * index + 3] - v.w * bs[6 * index + 4],
//			v.x * bs[6 * index + 1] + v.y * bs[6 * index + 3] - v.w * bs[6 * index + 5],
//			v.x * bs[6 * index + 2] + v.y * bs[6 * index + 4] + v.z * bs[6 * index + 5]
//		);
//}

__device__ static trivec4 wedge_bv(vec4 v, bivec4 b) {
	return
		trivec4(
			v.x * b.yz - v.y * b.xz + v.z * b.xy,
			v.x * b.yw - v.y * b.xw + v.w * b.xy,
			v.x * b.zw - v.z * b.xw + v.w * b.xz,
			v.x * b.zw - v.z * b.yw + v.w * b.yz);
}

__device__ static bivec4 wedge_v(vec4 u, vec4 v) {
	return
		bivec4(
			u.x * v.y - u.y * v.x,
			u.x * v.z - u.z * v.x,
			-u.w * v.x + u.x * v.w,
			u.y * v.z - u.z * v.y,
			-u.w * v.y + u.y * v.w,
			-u.w * v.z + u.z * v.w
		);
}

__device__ static void mul_bv(vec4 v, bivec4 b, vec4* a1, trivec4* a3)
{
	*a1 = left_contract_bv(v, b);
	*a3 = wedge_bv(v, b);
}

//bivec4
__device__ static vec4 dot_v(bivec4 b, vec4 v)
{
	return vec4(
		b.xw * v.w + b.xy * v.y + b.xz * v.z,
		-b.xy * v.x + b.yw * v.w + b.yz * v.z,
		-b.xz * v.x - b.yz * v.y + b.zw * v.w,
		-b.xw * v.x - b.yw * v.y - b.zw * v.z
	);
}

__device__ static trivec4 wedge_v(bivec4 b, vec4 v)
{
	return trivec4(
		b.xy * v.z - b.xz * v.y + b.yz * v.x,
		-b.xw * v.y + b.xy * v.w + b.yw * v.x,
		-b.xw * v.z + b.xz * v.w + b.zw * v.x,
		-b.yw * v.z + b.yz * v.w + b.zw * v.y
	);
}

__device__ static void mul_v(bivec4 b, vec4 v, vec4* a1, trivec4* a3)
{
	*a1 = dot_v(b, v);
	*a3 = wedge_v(b, v);
}

__device__ static float mul_bv_s(bivec4 b, bivec4 c)
{
	return
		-b.xy * c.xy
		- b.xz * c.xz
		- b.xw * c.xw
		- b.yz * c.yz
		- b.yw * c.yw
		- b.zw * c.zw;
}

__device__ static bivec4 mul_bv_b(bivec4 b, bivec4 c)
{
	return
		bivec4(
			-b.xw * c.yw - b.xz * c.yz + b.yw * c.xw + b.yz * c.xz,
			-b.xw * c.zw + b.xy * c.yz - b.yz * c.xy + b.zw * c.xw,
			b.xy * c.yw + b.xz * c.zw - b.yw * c.xy - b.zw * c.xz,
			-b.xy * c.xz + b.xz * c.xy - b.yw * c.zw + b.zw * c.yw,
			b.xw * c.xy - b.xy * c.xw + b.yz * c.zw - b.zw * c.yz,
			b.xw * c.xz - b.xz * c.xw + b.yw * c.yz - b.yz * c.yw
		);
}

__device__ static quadvec4 mul_bv_q(bivec4 b, bivec4 c)
{
	return
		quadvec4(
			b.xw * c.yz
			+ b.xy * c.zw
			- b.xz * c.yw
			- b.yw * c.xz
			+ b.yz * c.xw
			+ b.zw * c.xy);
}

__device__ static rotor4 mul_bv(bivec4 b, bivec4 c)
{
	return rotor4(mul_bv_s(b, c), mul_bv_b(b, c), mul_bv_q(b, c));
}

__device__ static void decompose(bivec4 b, bivec4* bPlus, bivec4* bMinus) {
	quadvec4 posHalfXYZW(quadvec4(0.5));
	quadvec4 negHalfXYZW(quadvec4(-0.5));
	*bPlus = b * 0.5f + posHalfXYZW.mul_bv(b);
	*bMinus = b * 0.5f + negHalfXYZW.mul_bv(b);
}


__device__ static rotor4 exp(bivec4 b) {
	bivec4 bPlus;
	bivec4 bMinus;
	decompose(b, &bPlus, &bMinus);
	float thetaPlus(2.f * sqrt(bPlus.xy * bPlus.xy + bPlus.xz * bPlus.xz + bPlus.xw * bPlus.xw));
	float thetaMinus(2.f * sqrt(bMinus.xy * bMinus.xy + bMinus.xz * bMinus.xz + bMinus.xw * bMinus.xw));

	float invThetaPlus(thetaPlus > 0.f ? 1.f / thetaPlus : 0.f);
	float invThetaMinus(thetaMinus > 0.f ? 1.f / thetaMinus : 0.f);

	bivec4 unitBPlus(bPlus * invThetaPlus);
	bivec4 unitBMinus(bMinus * invThetaMinus);
	float cosThetaPlus(cos(thetaPlus));
	float cosThetaMinus(cos(thetaMinus));
	return rotor4(
		0.5f * (cosThetaPlus + cosThetaMinus),
		unitBPlus * sin(thetaPlus) + unitBMinus * sin(thetaMinus),
		quadvec4(0.5f * (cosThetaPlus - cosThetaMinus))
	);
}
//trivec4
__device__ static vec4 right_contract_bv(trivec4 t, bivec4 b) {
	return vec4(
		-b.yw * t.xyw - b.yz * t.xyz - b.zw * t.xzw,
		b.xw * t.xyw + b.xz * t.xyz - b.zw * t.yzw,
		b.xw * t.xzw - b.xy * t.xyz + b.yw * t.yzw,
		-b.xy * t.xyw - b.xz * t.xzw - b.yz * t.yzw
	);
}

__device__ static vec4 mul_qv(trivec4 t, quadvec4 q)
{
	float xyzw(q.xyzw);
	return vec4(xyzw * t.yzw, -xyzw * t.xzw, xyzw * t.xyw, -xyzw * t.xyz);
}
//quadvec4
__device__ static trivec4 mul_v(quadvec4 q, vec4 v)
{
	return trivec4(q.xyzw * v.w, -q.xyzw * v.z, q.xyzw * v.y, -q.xyzw * v.x);
}

__device__ static bivec4 mul_bv(quadvec4 q, bivec4 b)
{
	float xyzw(q.xyzw);
	return bivec4(-b.zw * xyzw, b.yw * xyzw, -b.yz * xyzw, -b.xw * xyzw, b.xz * xyzw, -b.xy * xyzw);
}
//rotor4
__device__ static vec4 rotate(rotor4 r, vec4 v)
{
	// p = R v ~R. We do this in two steps:
	// Q = R v
	vec4 a1;
	trivec4 a3;
	mul_v(r.b, v, &a1, &a3);

	trivec4 b3(mul_v(r.q, v));
	vec4 q1(v * r.s + a1);
	trivec4 q3(a3 + b3);

	// p = Q ~R
	bivec4 bRev(r.b.reverse());
	vec4 p(q1 * r.s + left_contract_bv(q1, bRev) + right_contract_bv(q3, bRev) + mul_qv(q3, r.q));
	return p;
}

__device__ static mat4 to_matrix(rotor4 r)
{
	vec4 x = rotate(r, vec4(1.f, 0.f, 0.f, 0.f));
	vec4 y = rotate(r, vec4(0.f, 1.f, 0.f, 0.f));
	vec4 z = rotate(r, vec4(0.f, 0.f, 1.f, 0.f));
	vec4 w = rotate(r, vec4(0.f, 0.f, 0.f, 1.f));

	return {
		vec4(x.x, x.y, x.z, x.w),
		vec4(y.x, y.y, y.z, y.w),
		vec4(z.x, z.y, z.z, z.w),
		vec4(w.x, w.y, w.z, w.w)
	};
}

__device__ static void to_matrix(rotor4 r, vec4 scale4D, float* mats4, int index)
{
	vec4 x = rotate(r, vec4(1.f, 0.f, 0.f, 0.f));
	vec4 y = rotate(r, vec4(0.f, 1.f, 0.f, 0.f));
	vec4 z = rotate(r, vec4(0.f, 0.f, 1.f, 0.f));
	vec4 w = rotate(r, vec4(0.f, 0.f, 0.f, 1.f));
	mats4[16 * index] = x.x * scale4D.x, mats4[16 * index + 1] = x.y * scale4D.x, mats4[16 * index + 2] = x.z * scale4D.x, mats4[16 * index + 3] = x.w * scale4D.x,
	mats4[16 * index + 4] = y.x * scale4D.y, mats4[16 * index + 5] = y.y * scale4D.y, mats4[16 * index + 6] = y.z * scale4D.y, mats4[16 * index + 7] = y.w * scale4D.y,
	mats4[16 * index + 8] = z.x * scale4D.z, mats4[16 * index + 9] = z.y * scale4D.z, mats4[16 * index + 10] = z.z * scale4D.z, mats4[16 * index + 11] = z.w * scale4D.z,
	mats4[16 * index + 12] = w.x * scale4D.w, mats4[16 * index + 13] = w.y * scale4D.w, mats4[16 * index + 14] = w.z * scale4D.w, mats4[16 * index + 15] = w.w * scale4D.w;
}

__device__ static rotor4 mul_bv(rotor4 r, bivec4 c)
{
	return rotor4(mul_bv_s(r.b, c), c * r.s + mul_bv_b(r.b, c) + mul_bv(r.q, c), mul_bv_q(r.b, c));
}

__device__ static void decompose(rotor4 r, rotor4* rPlus, rotor4* rMinus)
{
	quadvec4 posHalfXYZW(quadvec4(0.5f));
	quadvec4 negHalfXYZW(quadvec4(-0.5f));

	*rPlus = rotor4(
		0.5f * (1.f + r.s + r.q.xyzw),
		r.b * 0.5f + posHalfXYZW.mul_bv(r.b),
		r.q * 0.5f + posHalfXYZW * r.s + negHalfXYZW
	);

	*rMinus = rotor4(
		0.5f * (1.f + r.s - r.q.xyzw),
		r.b * 0.5f + negHalfXYZW.mul_bv(r.b),
		r.q * 0.5f + negHalfXYZW * r.s + posHalfXYZW
	);
}

__device__ static rotor4 normalize(rotor4 r) {
	// we decompose into two isoclinic rotations, which are each equivalent
	// to a quaternion. Each quaternion component is normalised, and then we
	// recover the original rotor

	rotor4 rPlus;
	rotor4 rMinus;
	decompose(r, &rPlus, &rMinus);
	// get rid of the 1/2 (1 +- xyzw) components
	rPlus.s = rPlus.s - 0.5f;
	rMinus.s = rMinus.s - 0.5f;
	// we're going to overwrite the quadvector components since they should
	// be just +- the scalar components.

	float plusMag(2.f * sqrt(rPlus.s * rPlus.s + rPlus.b.xy * rPlus.b.xy + rPlus.b.xz * rPlus.b.xz + rPlus.b.xw * rPlus.b.xw));
	float minusMag(2.f * sqrt(rMinus.s * rMinus.s + rMinus.b.xy * rMinus.b.xy + rMinus.b.xz * rMinus.b.xz + rMinus.b.xw * rMinus.b.xw));

	if (plusMag > 0.f) {
		float invPlusMag(1.f / plusMag);
		float rPlusS(rPlus.s * invPlusMag + 0.5f);
		rPlus.set(rPlusS, rPlus.b * invPlusMag, quadvec4(rPlusS - 1.f));
		// readd 1/2 (1 - xyzw)
	}
	else {
		// TODO:
		// unimplemented!("{:?} has zero magnitude!", rPlus);
		rPlus.set();
	}

	if (minusMag > 0.f) {
		float invMinusMag(1.f / minusMag);
		float rMinusS(rMinus.s * invMinusMag + 0.5f);
		rMinus.set(rMinusS, rMinus.b * invMinusMag, quadvec4(1.f - rMinusS));
		// readd 1/2 (1 + xyzw)
	}
	else {
		// TODO
		// unimplemented!("{:?} has zero magnitude!", rMinus);
		rMinus.set();
	}

	return rPlus * rMinus;
}

__device__ static rotor4 update(rotor4 r, bivec4 delta) {
	return normalize(r * exp(delta * -0.5f));
}

__device__ static float length(bivec4 b)
{
	return sqrt(b.xy * b.xy + b.xz * b.xz + b.xw * b.xw + b.yz * b.yz + b.yw * b.yw + b.zw * b.zw);
}

__device__ static float mag(rotor4 r)
{
	return sqrt(r.s * r.s
		+ r.b.xy * r.b.xy
		+ r.b.xz * r.b.xz
		+ r.b.xw * r.b.xw
		+ r.b.yz * r.b.yz
		+ r.b.yw * r.b.yw
		+ r.b.zw * r.b.zw
		+ r.q.xyzw * r.q.xyzw);
}

__device__ static float weird_term(rotor4 r)
{
	return 2.f * (-r.b.xw * r.b.yz - r.b.xy * r.b.zw + r.b.xz * r.b.yw + r.q.xyzw * r.s);
}

__device__ static vec4 triple_cross_product(vec4 u, vec4 v, vec4 w)
{
	return mul_qv(wedge_v(wedge_v(vec4(u), vec4(v)), vec4(w)), quadvec4(1.f));
}
