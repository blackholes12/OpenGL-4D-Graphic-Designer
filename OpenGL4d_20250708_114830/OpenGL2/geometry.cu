#include "hip/hip_runtime.h"
#pragma once
#include "object.cu"

__device__ static vec2 cross2d(vec2 a2) {
	return vec2(a2.y, -a2.x);
}

__device__ static vec3 cross3d(vec3 a2, vec3 a3) {
	return vec3(
		a2.y * a3.z - a3.y * a2.z,
		a2.z * a3.x - a3.z * a2.x,
		a2.x * a3.y - a3.x * a2.y
	);
}

__device__ static vec4 cross4d(vec4 a2, vec4 a3, vec4 a4) {
	return vec4(
		a2.y * a3.z * a4.w - a2.y * a3.w * a4.z - a2.z * a3.y * a4.w + a2.z * a3.w * a4.y + a2.w * a3.y * a4.z - a2.w * a3.z * a4.y,
		-a2.x * a3.z * a4.w + a2.x * a3.w * a4.z + a2.z * a3.x * a4.w - a2.z * a3.w * a4.x - a2.w * a3.x * a4.z + a2.w * a3.z * a4.x,
		a2.x * a3.y * a4.w - a2.x * a3.w * a4.y - a2.y * a3.x * a4.w + a2.y * a3.w * a4.x + a2.w * a3.x * a4.y - a2.w * a3.y * a4.x,
		-a2.x * a3.y * a4.z + a2.x * a3.z * a4.y + a2.y * a3.x * a4.z - a2.y * a3.z * a4.x - a2.z * a3.x * a4.y + a2.z * a3.y * a4.x
	);
}

__device__ static void orthonormal_basis(vec4 normal4D, vec4* vec41, vec4* vec42, vec4* vec43)
{
	if (_d_abs(normal4D.x) < 1e-8f && _d_abs(normal4D.y) < 1e-8f && _d_abs(normal4D.z) < 1e-8f)
	{
		*vec41 = vec4(-normal4D.w, 0.f, 0.f, 0.f);
		*vec42 = vec4(0.f, 1.f, 0.f, 0.f);
		*vec43 = vec4(0.f, 0.f, 1.f, 0.f);
	}
	else if (_d_abs(normal4D.x) < 1e-8f && _d_abs(normal4D.y) < 1e-8f && _d_abs(normal4D.w) < 1e-8f)
	{
		*vec41 = vec4(normal4D.z, 0.f, 0.f, 0.f);
		*vec42 = vec4(0.f, 1.f, 0.f, 0.f);
		*vec43 = vec4(0.f, 0.f, 0.f, 1.f);
	}
	else if (_d_abs(normal4D.x) < 1e-8f && _d_abs(normal4D.z) < 1e-8f && _d_abs(normal4D.w) < 1e-8f)
	{
		*vec41 = vec4(-normal4D.y, 0.f, 0.f, 0.f);
		*vec42 = vec4(0.f, 0.f, 1.f, 0.f);
		*vec43 = vec4(0.f, 0.f, 0.f, 1.f);
	}
	else if (_d_abs(normal4D.y) < 1e-8f && _d_abs(normal4D.z) < 1e-8f && _d_abs(normal4D.w) < 1e-8f)
	{
		*vec41 = vec4(0.f, normal4D.x, 0.f, 0.f);
		*vec42 = vec4(0.f, 0.f, 1.f, 0.f);
		*vec43 = vec4(0.f, 0.f, 0.f, 1.f);
	}
	else
	{
		vec4 vA(vec4(0.f, 1.f, 0.f, 0.f));
		if (normal4D.y < 0.f)
			*vec41 = normalize(vA - normal4D * dot(vA, normal4D));
		else if (normal4D.y == 0.f)
			*vec41 = vA*-1.f;
		else
			*vec41 = normalize(vA - normal4D * dot(vA, normal4D))*-1.f;
		vec3 normalXZW(vec3(normal4D.x, normal4D.z, normal4D.w));
		if (_d_abs(normalXZW.y) < 1e-8f && _d_abs(normalXZW.z) < 1e-8f)
		{
			*vec42 = vec4(0.f, 0.f, 1.f, 0.f);
			*vec43 = vec4(0.f, 0.f, 0.f, 1.f);
		}
		else
		{
			vec3 v1(cross3d(vec3(1.f, 0.f, 0.f), normalXZW));
			*vec42 = normalize(vec4(v1.x, 0.f, v1.y, v1.z));
			vec3 v2(cross3d(v1, normalXZW));
			*vec43 = normalize(vec4(v2.x, 0.f, v2.y, v2.z));
		}
	}
}

__device__ static bool is_in_terrain_tetra(vec4 E, vec4 a, vec4 b, vec4 c, vec4 d)
{
	vec3 a2(vec3(a.x, a.z, a.w));
	vec3 b2(vec3(b.x, b.z, b.w));
	vec3 c2(vec3(c.x, c.z, c.w));
	vec3 d2(vec3(d.x, d.z, d.w));
	vec3 n1(cross3d(c2 - b2, b2 - a2));
	vec3 n2(cross3d(d2 - c2, c2 - a2));
	vec3 n3(cross3d(b2 - d2, d2 - a2));
	vec3 n4(cross3d(d2 - b2, b2 - c2));
	vec3 A(vec3(E.x, E.z, E.w) - a2);
	vec3 B(A);
	vec3 C(A);
	vec3 D(vec3(E.x, E.z, E.w) - b2);
	float dis1(dot(A, n1));
	float dis2(dot(B, n2));
	float dis3(dot(C, n3));
	float dis4(dot(D, n4));
	return dis1 < 0.f && dis2 < 0.f && dis3 < 0.f && dis4 < 0.f;
}

__device__ static bool is_in_tetra(vec4 E, vec4 normal4D, vec4 a, vec4 b, vec4 c, vec4 d)
{
	vec3 a2(vec3(a.x, a.z, a.w));
	vec3 b2(vec3(b.x, b.z, b.w));
	vec3 c2(vec3(c.x, c.z, c.w));
	vec3 d2(vec3(d.x, d.z, d.w));
	vec3 n1(cross3d(c2 - b2, b2 - a2));
	vec3 n2(cross3d(d2 - c2, c2 - a2));
	vec3 n3(cross3d(b2 - d2, d2 - a2));
	vec3 n4(cross3d(d2 - b2, b2 - c2));
	vec3 A(vec3(E.x, E.z, E.w) - a2);
	vec3 B(A);
	vec3 C(A);
	vec3 D(vec3(E.x, E.z, E.w) - b2);
	float dis1(dot(A, n1));
	float dis2(dot(B, n2));
	float dis3(dot(C, n3));
	float dis4(dot(D, n4));
	if (normal4D.y > 0.f)return dis1 <= 0.f && dis2 <= 0.f && dis3 <= 0.f && dis4 <= 0.f;
	else return dis1 >= 0.f && dis2 >= 0.f && dis3 >= 0.f && dis4 >= 0.f;
}

__device__ static vec4 get_normal_from_point_and_face(vec4 position4d, vec4 vertex4D1, vec4 vertex4D2, vec4 vertex4D3)
{
	vec4 ve12(normalize(vertex4D2 - vertex4D1));
	vec4 v13(vertex4D3 - vertex4D1);
	vec4 ne12(normalize(v13 - ve12 * dot(v13, ve12)));
	vec4 v1s(position4d - vertex4D1);
	float kv(dot(v1s, ve12));
	float kn(dot(v1s, ne12));
	vec4 closetPosToS(vertex4D1 + ve12 * kv + ne12 * kn);
	return normalize(position4d - closetPosToS);
}

__device__ static vec4 get_distance_and_normal_from_point_and_face(vec4 position4d, vec4 vertex4D1, vec4 vertex4D2, vec4 vertex4D3, float* distance)
{
	vec4 ve12(normalize(vertex4D2 - vertex4D1));
	vec4 v13(vertex4D3 - vertex4D1);
	vec4 ne12(normalize(v13 - ve12 * dot(v13, ve12)));
	vec4 v1s(position4d - vertex4D1);
	float kv(dot(v1s, ve12));
	float kn(dot(v1s, ne12));
	vec4 closetPosToS(vertex4D1 + ve12 * kv + ne12 * kn);
	vec4 normal(position4d - closetPosToS);
	*distance = length(normal);
	return normalize(normal);
}

__device__ static bool is_in_tetra2(vec4 E, vec4 a, vec4 b, vec4 c, vec4 d)
{
	vec4 n1(get_normal_from_point_and_face(d, a, b, c));
	vec4 n2(get_normal_from_point_and_face(a, b, c, d));
	vec4 n3(get_normal_from_point_and_face(b, c, d, a));
	vec4 n4(get_normal_from_point_and_face(c, d, a, b));
	return (dot(E - a, n1) >= 0.f && dot(E - b, n2) >= 0.f && dot(E - c, n3) >= 0.f && dot(E - d, n4) >= 0.f);
}

__device__ static bool is_in_tetra3(vec4 normal4D, vec4 E, vec4 a, vec4 b, vec4 c, vec4 d)
{
	vec4 contactPos0(E + normal4D * dot(normal4D, a - E));
	return is_in_tetra2(contactPos0, a, b, c, d);
}

__device__ static bool is_in_tetra4(vec4 E, vec4 normal4D, vec4 a, vec4 b, vec4 c, vec4 d)
{
	//bool isInTetra(false);
	if (normal4D.y != 0.f)
	{
		if (is_in_tetra(E, normal4D, a, b, c, d))
		{
			return true;
		}
	}
	else
	{
		if (is_in_tetra2(E, a, b, c, d))
		{
			return true;
		}
	}
	return false;
}

__device__ static vec4 get_point_from_edge_and_edge(vec4 vertex4DA1, vec4 vertex4DA2, vec4 vertex4DB1, vec4 vertex4DB2)
{
	vec4 dirB(normalize(vertex4DB2 - vertex4DB1));
	vec4 dirA1ToB1(vertex4DB1 - vertex4DA1);
	vec4 dirA2ToB2(vertex4DB2 - vertex4DA2);
	float distanceA1(length(dirA1ToB1 - dirB * dot(dirB, dirA1ToB1)));
	float distanceA2(length(dirA2ToB2 - dirB * dot(dirB, dirA2ToB2)));
	return (vertex4DA1 * distanceA2 + vertex4DA2 * distanceA1) / (distanceA1 + distanceA2);
}

__device__ static void get_edge_in_face_pos(vec4 normal4D, vec4 lineVertex1, vec4 lineVertex2, vec4 faceVertex1, vec4 faceVertex2, vec4 faceVertex3, vec4* contactPosA, vec4* contactPosB)
{
	vec4 shiftFromLineToFace(normal4D * dot(normal4D, faceVertex1 - lineVertex1));
	vec4 lineVertexShifted1(lineVertex1 + shiftFromLineToFace);
	vec4 lineVertexShifted2(lineVertex2 + shiftFromLineToFace);
	float distance1, distance2;
	get_distance_and_normal_from_point_and_face(lineVertexShifted1, faceVertex1, faceVertex2, faceVertex3, &distance1);
	get_distance_and_normal_from_point_and_face(lineVertexShifted2, faceVertex1, faceVertex2, faceVertex3, &distance2);
	vec4 intersectFacePoint;
	if (distance1 > 0.f)intersectFacePoint = mix(lineVertexShifted1, lineVertexShifted2, distance1 / (distance1 + distance2));
	else intersectFacePoint = (lineVertexShifted1 + lineVertexShifted2) / 2.f;
	*contactPosA = intersectFacePoint - shiftFromLineToFace;
	*contactPosB = intersectFacePoint;
}

__device__ static bool is_in_triangle(vec4 E, vec4 a, vec4 b, vec4 c)
{
	vec4 ca(normalize(c - a)), ab(normalize(a - b)), bc(normalize(b - c));
	vec4 n1(b - a - ca * dot(ca, b - a));
	vec4 n2(c - b - ab * dot(ab, c - b));
	vec4 n3(a - c - bc * dot(bc, a - c));
	return dot(E - a, n1) >= 0.f && dot(E - b, n2) >= 0.f && dot(E - c, n3) >= 0.f;
}

__device__ static bool is_in_square(vec4 E, vec4 a, vec4 b, vec4 c)
{
	return is_in_triangle(E, a, b, c) || is_in_triangle(E, c, c + a - b, a);
}

__device__ static bool is_edge_in_face(vec4 normal4D, vec4 lineVertex1, vec4 lineVertex2, vec4 faceVertex1, vec4 faceVertex2, vec4 faceVertex3, vec4* contactPos4D)
{
	vec4 shiftFromLineToFace(normal4D * dot(normal4D, faceVertex1 - lineVertex1));
	vec4 lineVertexShifted1(lineVertex1 + shiftFromLineToFace);
	vec4 lineVertexShifted2(lineVertex2 + shiftFromLineToFace);
	float distance1, distance2;
	vec4 normal1(get_distance_and_normal_from_point_and_face(lineVertexShifted1, faceVertex1, faceVertex2, faceVertex3, &distance1));
	vec4 normal2(get_distance_and_normal_from_point_and_face(lineVertexShifted2, faceVertex1, faceVertex2, faceVertex3, &distance2));
	vec4 intersectPoint(mix(lineVertexShifted1, lineVertexShifted2, distance1 / (distance1 + distance2)));
	*contactPos4D = intersectPoint - shiftFromLineToFace;
	return dot(normal1, normal2) < 0.f && is_in_triangle(intersectPoint, faceVertex1, faceVertex2, faceVertex3) && distance1 != distance2;
}

__device__ static bool is_edge_in_face(vec4 normal4D, vec4 lineVertex1, vec4 lineVertex2, vec4 faceVertex1, vec4 faceVertex2, vec4 faceVertex3)
{
	vec4 shiftFromLineToFace(normal4D * dot(normal4D, faceVertex1 - lineVertex1));
	vec4 lineVertexShifted1(lineVertex1 + shiftFromLineToFace);
	vec4 lineVertexShifted2(lineVertex2 + shiftFromLineToFace);
	float distance1, distance2;
	vec4 normal1(get_distance_and_normal_from_point_and_face(lineVertexShifted1, faceVertex1, faceVertex2, faceVertex3, &distance1));
	vec4 normal2(get_distance_and_normal_from_point_and_face(lineVertexShifted2, faceVertex1, faceVertex2, faceVertex3, &distance2));
	vec4 intersectPoint(distance1 + distance2 == 0.f ? (lineVertexShifted1 + lineVertexShifted2) / 2.f : mix(lineVertexShifted1, lineVertexShifted2, distance1 / (distance1 + distance2)));
	return dot(normal1, normal2) < 0.f && is_in_triangle(intersectPoint, faceVertex1, faceVertex2, faceVertex3);
}

__device__ static bool is_edge_in_square(vec4 normal4D, vec4 lineVertex1, vec4 lineVertex2, vec4 a, vec4 b, vec4 c)
{
	return is_edge_in_face(normal4D, lineVertex1, lineVertex2, a, b, c) || is_edge_in_face(normal4D, lineVertex1, lineVertex2, c, c + a - b, a);
}

__device__ static float get_tetra_height(vec4 E, vec4 A, vec4 B, vec4 C, vec4 D)
{
	vec4 normal4D(normalize(cross4d(A - B, A - C, A - D)));
	float l(dot(normal4D, E - A));
	float t(l / dot(normal4D*-1.f, vec4(0.f, -1.f, 0.f, 0.f)));
	return E.y - t;
}

__device__ static float get_tetra_height(vec4 E, vec4 normal4D, vec4 A, vec4 B, vec4 C, vec4 D)
{
	float l(dot(normal4D, E - A));
	float t(l / dot(normal4D*-1.f, vec4(0.f, -1.f, 0.f, 0.f)));
	return E.y - t;
}

__device__ static float get_tetra_height(vec4 E, vec4* normal4D, float* t, vec4 normal4D2, vec4 A, vec4 B, vec4 C, vec4 D)
{
	*normal4D = normal4D2;
	float dis(dot(normal4D2, E - A));
	*t = dis / dot(normal4D2*-1.f, vec4(0.f, -1.f, 0.f, 0.f));//distance/cos
	return E.y - *t;
}

__device__ static vec4 get_ray_detect_tetra_pos(vec4 cameraPos, vec4 forward, vec4 normal4D, vec4 A, vec4 B, vec4 C, vec4 D)
{
	float l(dot(normal4D, cameraPos - A));
	float t(l / dot(normal4D*1.f, forward));
	return cameraPos + forward * t;
}

__device__ static bool point_col_terrain_tetra4D(vec4 position4d,
	float* depth, vec4* normal4D, vec4* contactPos,
	vec4 normal4D0,
	vec4 vertex4D1,
	vec4 vertex4D2,
	vec4 vertex4D3,
	vec4 vertex4D4)
{
	bool isCol(false);
	float depth0(-dot(position4d - vertex4D1, normal4D0));
	if (depth0 > 0.f)
	{
		vec4 contactPos0(position4d + normal4D0 * depth0);
		if (is_in_tetra(contactPos0, *normal4D, vertex4D1, vertex4D2, vertex4D3, vertex4D4))
		{
			*depth = depth0;
			*normal4D = normal4D0;
			*contactPos = contactPos0;
			isCol = true;
		}
	}
	return isCol;
}

__device__ static bool sphere_col_terrain_tetra4D(vec4 spherePos4D, float radius,
	vec4 normal4D0,
	vec4 vertex4D1,
	vec4 vertex4D2,
	vec4 vertex4D3,
	vec4 vertex4D4,
	float* depth, vec4* normal4D, vec4* contactPosA, vec4* contactPosB)
{
	bool isCol(false);
	float depth0(radius - dot(spherePos4D - vertex4D1, normal4D0));
	if (depth0 > 0.f)
	{
		vec4 contactPosB0(spherePos4D - normal4D0 * (radius - depth0));
		vec4 contactPosA0(spherePos4D - normal4D0 * radius);
		if (is_in_tetra(contactPosB0, normal4D0, vertex4D1, vertex4D2, vertex4D3, vertex4D4))
		{
			*depth = depth0;
			*normal4D = normal4D0;
			*contactPosA = contactPosA0;
			*contactPosB = contactPosB0;
			isCol = true;
		}
	}
	return isCol;
}

__device__ static bool sphere_col_tetra4D(vec4 spherePos4D, float radius,
	vec4 normal4D0,
	vec4 vertex4D1,
	vec4 vertex4D2,
	vec4 vertex4D3,
	vec4 vertex4D4,
	float* depth, vec4* normal4D, vec4* contactPosA, vec4* contactPosB)
{
	bool isCol(false);
	float depth0(radius - dot(spherePos4D - vertex4D1, normal4D0));
	if (depth0 > 0.f && depth0 < radius)
	{
		vec4 contactPosB0(spherePos4D - normal4D0* (radius - depth0));
		vec4 contactPosA0(spherePos4D - normal4D0* radius);
		if (is_in_tetra4(contactPosB0, normal4D0, vertex4D1, vertex4D2, vertex4D3, vertex4D4))
		{
			*depth = depth0;
			*normal4D = normal4D0;
			*contactPosA = contactPosA0;
			*contactPosB = contactPosB0;
			isCol = true;
		}
	}
	return isCol;
}

__device__ static bool sphere_col_face4D(vec4 spherePos4D, float radius, vec4 vertex4D1, vec4 vertex4D2, vec4 vertex4D3, float* depth, vec4* normal4D, vec4* contactPosA, vec4* contactPosB)
{
	bool isCol(false);
	vec4 normal4D0(get_normal_from_point_and_face(spherePos4D, vertex4D1, vertex4D2, vertex4D3));
	float depth0(radius - dot(spherePos4D - vertex4D1, normal4D0));
	if (depth0 > 0.f)
	{
		vec4 contactPosB0(spherePos4D - normal4D0* (radius - depth0));
		vec4 contactPosA0(spherePos4D - normal4D0* radius);
		if (is_in_triangle(contactPosB0, vertex4D1, vertex4D2, vertex4D3))
		{
			*depth = depth0;
			*normal4D = normal4D0;
			*contactPosA = contactPosA0;
			*contactPosB = contactPosB0;
			isCol = true;
		}
	}
	return isCol;
}

__device__ static bool sphere_col_line4D(vec4 spherePos4D, float radius, vec4 vertex4D1, vec4 vertex4D2, float* depth, vec4* normal4D, vec4* contactPosA, vec4* contactPosB)
{
	bool isCol(false);
	vec4 normal4D0(normalize(spherePos4D - vertex4D1 - normalize(vertex4D2 - vertex4D1) * dot(normalize(vertex4D2 - vertex4D1), spherePos4D - vertex4D1)));
	float depth0(radius - dot(spherePos4D - vertex4D1, normal4D0));
	if (depth0 > 0.f)
	{
		vec4 contactPosB0(spherePos4D - normal4D0* (radius - depth0));
		vec4 contactPosA0(spherePos4D - normal4D0* radius);
		if (dot(vertex4D1 - contactPosB0, vertex4D2 - contactPosB0) < 0.f)
		{
			*depth = depth0;
			*normal4D = normal4D0;
			*contactPosA = contactPosA0;
			*contactPosB = contactPosB0;
			isCol = true;
		}
	}
	return isCol;
}

__device__ static bool sphere_col_point4D(vec4 spherePos4D, float radius, vec4 vertex4D, float* depth, vec4* normal4D, vec4* contacts)
{
	bool isCol(false);
	vec4 normal4D0(normalize(spherePos4D - vertex4D));
	float depth0(radius - dot(spherePos4D - vertex4D, normal4D0));
	if (depth0 > 0.f)
	{
		vec4 contacts0(spherePos4D - normal4D0* radius);
		*depth = depth0;
		*normal4D = normal4D0;
		*contacts = contacts0;
		isCol = true;
	}
	return isCol;
}

__device__ static bool get_ray_detect_tetra_distance(vec4 cameraPos, vec4 forward, vec4 normal4D, vec4 A, vec4 B, vec4 C, vec4 D, float* t)
{
	bool isCol(false);
	if (dot(forward, normal4D) < 0.f)
	{
		float l(dot(normal4D, cameraPos - A));
		float t2(-l / dot(normal4D, forward));
		if (t2 < *t && t2 > 0.f)
		{
			vec4 hitPos0(cameraPos + forward * t2);
			if (is_in_tetra4(hitPos0, normal4D, A, B, C, D))
			{
				//std::cout<<"is in tetra"<<"\n";
				*t = t2;
				isCol = true;
			}
		}
	}
	return isCol;
}

//x=0,y=1,z=2,w=3,-x=4,-y=5,-z=6,-w=7
//a direction
__device__ static bool col_plane4D(vec4 startPos4D, vec4 endPos4D, vec4 planePos4D, vec4 planeScale4D, float* t, vec4* hitNormal, const unsigned int a)
{
	bool isCol(false);
	unsigned int a2;
	float sign;
	if (a > 3)a2 = a - 4, sign = -1.f;
	else a2 = a, sign = 1.f;
	vec4 normal4D(vec4(0.f));
	normal4D.value(a2, a > 3 ? -1.f : 1.f);
	if ((startPos4D[a2] - endPos4D[a2]) * normal4D[a2] > 0.f)
	{
		if ((startPos4D[a2] - planePos4D[a2]) * (endPos4D[a2] - planePos4D[a2]) < 0.f)
		{
			unsigned int b2, c2, d2;
			if (a2 == 0)b2 = 1, c2 = 2, d2 = 3;
			else if (a2 == 1)b2 = 0, c2 = 2, d2 = 3;
			else if (a2 == 2)b2 = 0, c2 = 1, d2 = 3;
			else if (a2 == 3)b2 = 0, c2 = 1, d2 = 2;
			vec4 rcolPos4D(startPos4D + (endPos4D - startPos4D) * (planePos4D[a2] - startPos4D[a2]) / (endPos4D[a2] - startPos4D[a2]) - planePos4D);
			isCol =
				_d_abs(rcolPos4D[b2]) <= planeScale4D[b2] / 2.f &&
				_d_abs(rcolPos4D[c2]) <= planeScale4D[c2] / 2.f &&
				_d_abs(rcolPos4D[d2]) <= planeScale4D[d2] / 2.f;
			if (isCol)
			{
				//std::cout << "hit" << "\n";
				vec4 hitPos(rcolPos4D + planePos4D);
				float t2(length(startPos4D - hitPos));
				if (t2 < *t)
				{
					//std::cout << "length of ray=" << length(startPos4D - hitPos) << "\n";
					*t = t2;
					*hitNormal = normal4D;
				}
			}
		}
	}
	return isCol;
}

__device__ static bool line_sigment_detect_box4D(vec4 startPos4D, vec4 endPos4D, vec4 scale4D, float* t, vec4* hitNormal)
{
	unsigned int a2;
	float sign;
	for (int i(0); i < 8; i++)
	{
		if (i > 3)a2 = i - 4, sign = -1.f;
		else a2 = i, sign = 1.f;
		vec4 planePos4D(vec4(0.f));
		planePos4D.value(a2, sign * scale4D[a2] / 2.f);
		if (col_plane4D(startPos4D, endPos4D, planePos4D, scale4D, t, hitNormal, i))
		{
			return true;
		}
	}
	return false;
}

__device__ static void vertices_used(float distance, vec4 vertexPos4D, vec4 normal0, vec4 objectPosition4D, vec4* vertices4D, int* sizeOfVertices)
{
	if (*sizeOfVertices < 16) {
		if (distance < 0.01f)
		{
			bool isAdd(true);
			for (int j(0); j < *sizeOfVertices; j++)
			{
				if (vertices4D[j] == vertexPos4D)
				{
					isAdd = false;
				}
			}
			if (isAdd)
				vertices4D[*sizeOfVertices] = vertexPos4D, * sizeOfVertices = *sizeOfVertices + 1;
		}
	}
}

__device__ static void edges_used(vec2 distances, _d_Edge4D edge4D, vec4 normal0, vec4 objectPosition4D, _d_Edge4D* edges4D, int* sizeOfEdges)
{
	if (*sizeOfEdges < 32) {
		if (distances[0] < 0.01f &&
			distances[1] < 0.01f)
		{
			bool isAdd(true);
			for (int j(0); j < *sizeOfEdges; j++)
			{
				if ((edges4D[j].vertices4D[0] == edge4D.vertices4D[0] && edges4D[j].vertices4D[1] == edge4D.vertices4D[1]) ||
					(edges4D[j].vertices4D[0] == edge4D.vertices4D[1] && edges4D[j].vertices4D[1] == edge4D.vertices4D[0]))
				{
					isAdd = false;
				}
			}
			if (isAdd)
				edges4D[*sizeOfEdges] = edge4D; *sizeOfEdges = *sizeOfEdges + 1;
		}
	}
}

__device__ static void faces_used(vec3 distances, _d_Face4D face4D, vec4 normal0, vec4 objectPosition4D, _d_Face4D* faces4D, int* sizeOfFaces)
{
	if (*sizeOfFaces < 24) {
		if (distances[0] < 0.01f &&
			distances[1] < 0.01f &&
			distances[2] < 0.01f)
		{
			bool isAdd(true);
			for (int j(0); j < *sizeOfFaces; j++)
			{
				if ((faces4D[j].vertices4D[0] == face4D.vertices4D[0] && faces4D[j].vertices4D[1] == face4D.vertices4D[1] && faces4D[j].vertices4D[2] == face4D.vertices4D[2]) ||
					(faces4D[j].vertices4D[0] == face4D.vertices4D[0] && faces4D[j].vertices4D[1] == face4D.vertices4D[2] && faces4D[j].vertices4D[2] == face4D.vertices4D[1]) ||
					(faces4D[j].vertices4D[0] == face4D.vertices4D[1] && faces4D[j].vertices4D[1] == face4D.vertices4D[0] && faces4D[j].vertices4D[2] == face4D.vertices4D[2]) ||
					(faces4D[j].vertices4D[0] == face4D.vertices4D[1] && faces4D[j].vertices4D[1] == face4D.vertices4D[2] && faces4D[j].vertices4D[2] == face4D.vertices4D[0]) ||
					(faces4D[j].vertices4D[0] == face4D.vertices4D[2] && faces4D[j].vertices4D[1] == face4D.vertices4D[0] && faces4D[j].vertices4D[2] == face4D.vertices4D[1]) ||
					(faces4D[j].vertices4D[0] == face4D.vertices4D[2] && faces4D[j].vertices4D[1] == face4D.vertices4D[1] && faces4D[j].vertices4D[2] == face4D.vertices4D[0]))
				{
					isAdd = false;
				}
			}
			if (isAdd)
				faces4D[*sizeOfFaces] = face4D; *sizeOfFaces = *sizeOfFaces + 1;
		}
	}
}

__device__ static void tetras_used(vec4 distances, _d_Tetra4D tetra4D, vec4 normal0, vec4 objectPosition4D, _d_Tetra4D* tetras4D, int* sizeOfTetras)
{
	if (*sizeOfTetras < 16) {
		if (distances[0] < 0.01f &&
			distances[1] < 0.01f &&
			distances[2] < 0.01f &&
			distances[3] < 0.01f)
		{
			tetras4D[*sizeOfTetras] = tetra4D; *sizeOfTetras = *sizeOfTetras + 1;
		}
	}
}
