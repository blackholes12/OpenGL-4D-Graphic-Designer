#include "hip/hip_runtime.h"
#pragma once
#include "Game.h"
#include "phyFunction.cu"
__device__ static _d_RigidBody4D copy_data_to_rigid_body(
	int index,
	bool isRigidBody,
	float* positions4D,
	float* rotations4D,
	float* scales4D,
	bool* isGravitys,
	bool* isStatics,
	bool* isSleeps,
	bool* isContinueSleeps,
	int* sizesOfForces,
	int* forceStartIndices,
	float* masss,
	float* momentInertiaScalars,
	float* restitutions,
	float* mus,
	float* massCenters,
	float* velocitys4D,
	float* angularVelocitys4D,
	int* colTypes,
	int* primitiveIds
)
{
	if (isRigidBody)
	{
		vec4 position4d = vec4(positions4D[4 * index], positions4D[4 * index + 1], positions4D[4 * index + 2], positions4D[4 * index + 3]);
		rotor4 rotation4D = rotor4(rotations4D[8 * index], bivec4(rotations4D[8 * index + 1], rotations4D[8 * index + 2], rotations4D[8 * index + 3], rotations4D[8 * index + 4], rotations4D[8 * index + 5], rotations4D[8 * index + 6]), quadvec4(rotations4D[8 * index + 7]));
		vec4 scale4D = vec4(scales4D[4 * index], scales4D[4 * index + 1], scales4D[4 * index + 2], scales4D[4 * index + 3]);
		bool isGravity = isGravitys[index];
		bool isStatic = isStatics[index];
		bool isSleep = isSleeps[index];
		bool isContinueSleep = isContinueSleeps[index];
		int sizeOfForces = sizesOfForces[index];
		int forcesStartIndex = forceStartIndices[index];
		float mass = masss[index];
		float momentInertiaScalar = momentInertiaScalars[index];
		float restitution = restitutions[index];
		float mu = mus[index];
		vec4 massCenter = vec4(massCenters[4 * index], massCenters[4 * index + 1], massCenters[4 * index + 2], massCenters[4 * index + 3]);
		vec4 velocity4d = vec4(velocitys4D[4 * index], velocitys4D[4 * index + 1], velocitys4D[4 * index + 2], velocitys4D[4 * index + 3]);
		bivec4 angularVelocity4D = bivec4(angularVelocitys4D[6 * index], angularVelocitys4D[6 * index + 1], angularVelocitys4D[6 * index + 2], angularVelocitys4D[6 * index + 3], angularVelocitys4D[6 * index + 4], angularVelocitys4D[6 * index + 5]);
		int colType = colTypes[index];
		int primitiveId = primitiveIds[index];
		return _d_RigidBody4D(position4d, rotation4D, scale4D, isGravity, isStatic, isSleep, isContinueSleep, sizeOfForces, forcesStartIndex, mass, momentInertiaScalar, restitution, mu, massCenter, velocity4d, angularVelocity4D, colType, primitiveId);
	}
	else
	{
		vec4 position4d = vec4(positions4D[4 * index], positions4D[4 * index + 1], positions4D[4 * index + 2], positions4D[4 * index + 3]);
		rotor4 rotation4D = rotor4(rotations4D[8 * index], bivec4(rotations4D[8 * index + 1], rotations4D[8 * index + 2], rotations4D[8 * index + 3], rotations4D[8 * index + 4], rotations4D[8 * index + 5], rotations4D[8 * index + 6]), quadvec4(rotations4D[8 * index + 7]));
		vec4 scale4D = vec4(scales4D[4 * index], scales4D[4 * index + 1], scales4D[4 * index + 2], scales4D[4 * index + 3]);
		float restitution = restitutions[index];
		float mu = mus[index];
		int colType = colTypes[index];
		int primitiveId = primitiveIds[index];
		return _d_RigidBody4D(position4d, rotation4D, scale4D, restitution, mu, colType, primitiveId);
	}
}

__device__ static _d_RigidBody4D copy_data_to_wall(
	int index,
	float* positions4D,
	float* rotations4D,
	float* scales4D,
	float* restitutions,
	float* mus,
	int* colTypes,
	int* primitiveIds
)
{
	vec4 position4d = vec4(positions4D[4 * index], positions4D[4 * index + 1], positions4D[4 * index + 2], positions4D[4 * index + 3]);
	rotor4 rotation4D = rotor4(rotations4D[8 * index], bivec4(rotations4D[8 * index + 1], rotations4D[8 * index + 2], rotations4D[8 * index + 3], rotations4D[8 * index + 4], rotations4D[8 * index + 5], rotations4D[8 * index + 6]), quadvec4(rotations4D[8 * index + 7]));
	vec4 scale4D = vec4(scales4D[4 * index], scales4D[4 * index + 1], scales4D[4 * index + 2], scales4D[4 * index + 3]);
	float restitution = restitutions[index];
	float mu = mus[index];
	int colType = colTypes[index];
	int primitiveId = primitiveIds[index];
	return _d_RigidBody4D(position4d, rotation4D, scale4D, restitution, mu, colType, primitiveId);
}

__device__ static void move_data_from_rigid_body(
	int index,
	_d_RigidBody4D a,
	float* positions4D,
	float* rotations4D,
	bool* isSleeps,
	bool* isContinueSleeps,
	float* velocitys4D,
	float* angularVelocitys4D
)
{
	positions4D[4 * index] = a.position4d.x, positions4D[4 * index + 1] = a.position4d.y, positions4D[4 * index + 2] = a.position4d.z, positions4D[4 * index + 3] = a.position4d.w;
	rotations4D[8 * index] = a.rotation4D.s, rotations4D[8 * index + 1] = a.rotation4D.b.xy, rotations4D[8 * index + 2] = a.rotation4D.b.xz, rotations4D[8 * index + 3] = a.rotation4D.b.xw, rotations4D[8 * index + 4] = a.rotation4D.b.yz, rotations4D[8 * index + 5] = a.rotation4D.b.yw, rotations4D[8 * index + 6] = a.rotation4D.b.zw, rotations4D[8 * index + 7] = a.rotation4D.q.xyzw;
	isSleeps[index] = a.isSleep;
	isContinueSleeps[index] = a.isContinueSleep;
	velocitys4D[4 * index] = a.velocity4d.x, velocitys4D[4 * index + 1] = a.velocity4d.y, velocitys4D[4 * index + 2] = a.velocity4d.z, velocitys4D[4 * index + 3] = a.velocity4d.w;
	angularVelocitys4D[6 * index] = a.angularVelocity4D.xy, angularVelocitys4D[6 * index + 1] = a.angularVelocity4D.xz, angularVelocitys4D[6 * index + 2] = a.angularVelocity4D.xw, angularVelocitys4D[6 * index + 3] = a.angularVelocity4D.yz, angularVelocitys4D[6 * index + 4] = a.angularVelocity4D.yw, angularVelocitys4D[6 * index + 5] = a.angularVelocity4D.zw;
}

__device__ static void move_velocity_data_from_rigid_body(
	int index,
	_d_RigidBody4D a,
	float* velocitys4D,
	float* angularVelocitys4D
)
{
	velocitys4D[4 * index] = a.velocity4d.x, velocitys4D[4 * index + 1] = a.velocity4d.y, velocitys4D[4 * index + 2] = a.velocity4d.z, velocitys4D[4 * index + 3] = a.velocity4d.w;
	angularVelocitys4D[6 * index] = a.angularVelocity4D.xy, angularVelocitys4D[6 * index + 1] = a.angularVelocity4D.xz, angularVelocitys4D[6 * index + 2] = a.angularVelocity4D.xw, angularVelocitys4D[6 * index + 3] = a.angularVelocity4D.yz, angularVelocitys4D[6 * index + 4] = a.angularVelocity4D.yw, angularVelocitys4D[6 * index + 5] = a.angularVelocity4D.zw;
}

__device__ static _d_Constraint copy_data_to_constraint(
	int index,
	int* constraintsVecIntArray,
	float* constraintsVecFloatArray
)
{
	int arrayIndexF = 50 * index;
	return{
	    constraintsVecIntArray[2 * index],
		constraintsVecIntArray[2 * index + 1],
		{{vec4(constraintsVecFloatArray[arrayIndexF],
		constraintsVecFloatArray[arrayIndexF + 1],
		constraintsVecFloatArray[arrayIndexF + 2],
		constraintsVecFloatArray[arrayIndexF + 3]),
		vec4(constraintsVecFloatArray[arrayIndexF + 4],
		constraintsVecFloatArray[arrayIndexF + 5],
		constraintsVecFloatArray[arrayIndexF + 6],
		constraintsVecFloatArray[arrayIndexF + 7]),
		vec4(constraintsVecFloatArray[arrayIndexF + 8],
		constraintsVecFloatArray[arrayIndexF + 9],
		constraintsVecFloatArray[arrayIndexF + 10],
		constraintsVecFloatArray[arrayIndexF + 11]),
		vec4(constraintsVecFloatArray[arrayIndexF + 12],
		constraintsVecFloatArray[arrayIndexF + 13],
		constraintsVecFloatArray[arrayIndexF + 14],
		constraintsVecFloatArray[arrayIndexF + 15])},
		{vec4(constraintsVecFloatArray[arrayIndexF + 16],
		constraintsVecFloatArray[arrayIndexF + 17],
		constraintsVecFloatArray[arrayIndexF + 18],
		constraintsVecFloatArray[arrayIndexF + 19]),
		vec4(constraintsVecFloatArray[arrayIndexF + 20],
		constraintsVecFloatArray[arrayIndexF + 21],
		constraintsVecFloatArray[arrayIndexF + 22],
		constraintsVecFloatArray[arrayIndexF + 23]),
		constraintsVecFloatArray[arrayIndexF + 24],
		{vec3(constraintsVecFloatArray[arrayIndexF + 25],
		constraintsVecFloatArray[arrayIndexF + 26],
		constraintsVecFloatArray[arrayIndexF + 27]),
		vec3(constraintsVecFloatArray[arrayIndexF + 28],
		constraintsVecFloatArray[arrayIndexF + 29],
		constraintsVecFloatArray[arrayIndexF + 30]),
		vec3(constraintsVecFloatArray[arrayIndexF + 31],
		constraintsVecFloatArray[arrayIndexF + 32],
		constraintsVecFloatArray[arrayIndexF + 33]),
		vec3(constraintsVecFloatArray[arrayIndexF + 34],
		constraintsVecFloatArray[arrayIndexF + 35],
		constraintsVecFloatArray[arrayIndexF + 36])},
		vec4(constraintsVecFloatArray[arrayIndexF + 37],
		constraintsVecFloatArray[arrayIndexF + 38],
		constraintsVecFloatArray[arrayIndexF + 39],
		constraintsVecFloatArray[arrayIndexF + 40])},
		constraintsVecFloatArray[arrayIndexF + 41]},
		vec4(constraintsVecFloatArray[arrayIndexF + 42],
		constraintsVecFloatArray[arrayIndexF + 43],
		constraintsVecFloatArray[arrayIndexF + 44],
		constraintsVecFloatArray[arrayIndexF + 45]),
		constraintsVecFloatArray[arrayIndexF + 46],
		constraintsVecFloatArray[arrayIndexF + 47],
		constraintsVecFloatArray[arrayIndexF + 48],
		constraintsVecFloatArray[arrayIndexF + 49]
	};
}

__device__ static void move_data_from_constraint(
	int index,
	_d_Constraint constraint,
	int* constraintsVecIntArray,
	float* constraintsVecFloatArray
)
{
	constraintsVecIntArray[2 * index] = constraint.index1;
	constraintsVecIntArray[2 * index + 1] = constraint.index2;
	int arrayIndexF = 50 * index;
	constraintsVecFloatArray[arrayIndexF] = constraint.collisionConstraint.orthonormalBasis[0].x;
	constraintsVecFloatArray[arrayIndexF + 1] = constraint.collisionConstraint.orthonormalBasis[0].y;
	constraintsVecFloatArray[arrayIndexF + 2] = constraint.collisionConstraint.orthonormalBasis[0].z;
	constraintsVecFloatArray[arrayIndexF + 3] = constraint.collisionConstraint.orthonormalBasis[0].w;
	constraintsVecFloatArray[arrayIndexF + 4] = constraint.collisionConstraint.orthonormalBasis[1].x;
	constraintsVecFloatArray[arrayIndexF + 5] = constraint.collisionConstraint.orthonormalBasis[1].y;
	constraintsVecFloatArray[arrayIndexF + 6] = constraint.collisionConstraint.orthonormalBasis[1].z;
	constraintsVecFloatArray[arrayIndexF + 7] = constraint.collisionConstraint.orthonormalBasis[1].w;
	constraintsVecFloatArray[arrayIndexF + 8] = constraint.collisionConstraint.orthonormalBasis[2].x;
	constraintsVecFloatArray[arrayIndexF + 9] = constraint.collisionConstraint.orthonormalBasis[2].y;
	constraintsVecFloatArray[arrayIndexF + 10] = constraint.collisionConstraint.orthonormalBasis[2].z;
	constraintsVecFloatArray[arrayIndexF + 11] = constraint.collisionConstraint.orthonormalBasis[2].w;
	constraintsVecFloatArray[arrayIndexF + 12] = constraint.collisionConstraint.orthonormalBasis[3].x;
	constraintsVecFloatArray[arrayIndexF + 13] = constraint.collisionConstraint.orthonormalBasis[3].y;
	constraintsVecFloatArray[arrayIndexF + 14] = constraint.collisionConstraint.orthonormalBasis[3].z;
	constraintsVecFloatArray[arrayIndexF + 15] = constraint.collisionConstraint.orthonormalBasis[3].w;
	constraintsVecFloatArray[arrayIndexF + 16] = constraint.collisionConstraint.contacts.contactPosA.x;
	constraintsVecFloatArray[arrayIndexF + 17] = constraint.collisionConstraint.contacts.contactPosA.y;
	constraintsVecFloatArray[arrayIndexF + 18] = constraint.collisionConstraint.contacts.contactPosA.z;
	constraintsVecFloatArray[arrayIndexF + 19] = constraint.collisionConstraint.contacts.contactPosA.w;
	constraintsVecFloatArray[arrayIndexF + 20] = constraint.collisionConstraint.contacts.contactPosB.x;
	constraintsVecFloatArray[arrayIndexF + 21] = constraint.collisionConstraint.contacts.contactPosB.y;
	constraintsVecFloatArray[arrayIndexF + 22] = constraint.collisionConstraint.contacts.contactPosB.z;
	constraintsVecFloatArray[arrayIndexF + 23] = constraint.collisionConstraint.contacts.contactPosB.w;
	constraintsVecFloatArray[arrayIndexF + 24] = constraint.collisionConstraint.contacts.bias;
	constraintsVecFloatArray[arrayIndexF + 25] = constraint.collisionConstraint.contacts.orthonormalBasisInv[0].x;
	constraintsVecFloatArray[arrayIndexF + 26] = constraint.collisionConstraint.contacts.orthonormalBasisInv[0].y;
	constraintsVecFloatArray[arrayIndexF + 27] = constraint.collisionConstraint.contacts.orthonormalBasisInv[0].z;
	constraintsVecFloatArray[arrayIndexF + 28] = constraint.collisionConstraint.contacts.orthonormalBasisInv[1].x;
	constraintsVecFloatArray[arrayIndexF + 29] = constraint.collisionConstraint.contacts.orthonormalBasisInv[1].y;
	constraintsVecFloatArray[arrayIndexF + 30] = constraint.collisionConstraint.contacts.orthonormalBasisInv[1].z;
	constraintsVecFloatArray[arrayIndexF + 31] = constraint.collisionConstraint.contacts.orthonormalBasisInv[2].x;
	constraintsVecFloatArray[arrayIndexF + 32] = constraint.collisionConstraint.contacts.orthonormalBasisInv[2].y;
	constraintsVecFloatArray[arrayIndexF + 33] = constraint.collisionConstraint.contacts.orthonormalBasisInv[2].z;
	constraintsVecFloatArray[arrayIndexF + 34] = constraint.collisionConstraint.contacts.orthonormalBasisInv[3].x;
	constraintsVecFloatArray[arrayIndexF + 35] = constraint.collisionConstraint.contacts.orthonormalBasisInv[3].y;
	constraintsVecFloatArray[arrayIndexF + 36] = constraint.collisionConstraint.contacts.orthonormalBasisInv[3].z;
	constraintsVecFloatArray[arrayIndexF + 37] = constraint.collisionConstraint.contacts.orthonormalBasisImpulse.x;
	constraintsVecFloatArray[arrayIndexF + 38] = constraint.collisionConstraint.contacts.orthonormalBasisImpulse.y;
	constraintsVecFloatArray[arrayIndexF + 39] = constraint.collisionConstraint.contacts.orthonormalBasisImpulse.z;
	constraintsVecFloatArray[arrayIndexF + 40] = constraint.collisionConstraint.contacts.orthonormalBasisImpulse.w;
	constraintsVecFloatArray[arrayIndexF + 41] = constraint.collisionConstraint.mu;
	constraintsVecFloatArray[arrayIndexF + 42] = constraint.impulse.x;
	constraintsVecFloatArray[arrayIndexF + 43] = constraint.impulse.y;
	constraintsVecFloatArray[arrayIndexF + 44] = constraint.impulse.z;
	constraintsVecFloatArray[arrayIndexF + 45] = constraint.impulse.w;
	constraintsVecFloatArray[arrayIndexF + 46] = constraint.momentInertiaScalarA;
	constraintsVecFloatArray[arrayIndexF + 47] = constraint.momentInertiaScalarB;
	constraintsVecFloatArray[arrayIndexF + 48] = constraint.impulseMagAdd;
	constraintsVecFloatArray[arrayIndexF + 49] = constraint.maxImpulseAdd;
}

//__global__ static void apply_forces(
//	float* positions4D, 
//	float* rotations4D, 
//	float* scales4D, 
//	bool* isGravitys,
//	bool* isStatics,
//	bool* isSleeps,
//	bool* isContinueSleeps,
//	float* forceSystems4D,
//	int* sizesOfForces,
//	int* forceStartIndices,
//	float* masss, 
//	float* momentInertiaScalars,
//	float* restitutions,
//	float* mus,
//	float* massCenters,
//	float* velocitys4D,
//	float* angularVelocitys4D,
//	int* colTypes,
//	int* primitiveIds,
//	float dt,
//	int sizeOfRigidBodies
//)
//{
//	int index = blockDim.x * blockIdx.x + threadIdx.x;
//
//}

__global__ static void detect_collision(
	float* positions4D,
	float* rotations4D,
	float* scales4D,
	bool* isGravitys,
	bool* isStatics,
	bool* isSleeps,
	bool* isContinueSleeps,
	int* sizesOfForces,
	int* forceStartIndices,
	float* masss,
	float* momentInertiaScalars,
	float* restitutions,
	float* mus,
	float* massCenters,
	float* velocitys4D,
	float* angularVelocitys4D,
	int* colTypes,
	float* vertices4DsVec,
	int* sizesOfVertices4D,
	int* verticesStartIndices,
	int* primitiveIds,
	int* pairsVec,
	int* sizesOfPairs,
	int* pairsStartIndices,
	int* collisionsVecIntArray,
	float* collisionsVecFloatArray,
	int* sizesOfCollisionsVec,
	float* constraintsVecFloatArray,
	int* pPairsVec,
	int* pSizesOfPairs,
	int* pPairsStartIndices,
	int* pSizesOfCollisionsVec,
	size_t pSizeOfPairs,
	float dt,
	int sizeOfRigidBodies
)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int index1 = pairsVec[2 * index];
	int index2 = pairsVec[2 * index + 1];
	_d_Collision collisions[32];
	sizesOfCollisionsVec[index] = 0;
	_d_RigidBody4D a = copy_data_to_rigid_body(index1, true, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
	_d_RigidBody4D b = copy_data_to_rigid_body(_d_max(index2, 0), index2 < sizeOfRigidBodies, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
	if (index2 != -1)
	{
		if (!isStatics[index1])
		{
			detect_rigidbody_or_wall(index1, index2, a, b, vertices4DsVec, sizesOfVertices4D, verticesStartIndices, collisions, &sizesOfCollisionsVec[index]);
		}
		else if (isStatics[index1] && !isStatics[index2] && index2 < sizeOfRigidBodies)
		{
			detect_rigidbody_or_wall(index2, index1, b, a, vertices4DsVec, sizesOfVertices4D, verticesStartIndices, collisions, &sizesOfCollisionsVec[index]);
		}
	}
	else
	{
		if (!isStatics[index1])
		{
			detect_rigidbody_edge(index1, a, vertices4DsVec, sizesOfVertices4D, verticesStartIndices, collisions, &sizesOfCollisionsVec[index]);
		}
	}

	//if (sizesOfCollisionsVec[index] > 0)
	//{
	//	for (int j(0); j < 20; j++)
	//	{
	//		for (int i(0); i < sizesOfCollisionsVec[index]; i++)
	//		{
	//			a.position4d = vec4(positions4D[4 * index1], positions4D[4 * index1 + 1], positions4D[4 * index1 + 2], positions4D[4 * index1 + 3]);
	//			a.rotation4D = rotor4(rotations4D[8 * index1], bivec4(rotations4D[8 * index1 + 1], rotations4D[8 * index1 + 2], rotations4D[8 * index1 + 3], rotations4D[8 * index1 + 4], rotations4D[8 * index1 + 5], rotations4D[8 * index1 + 6]), quadvec4(rotations4D[8 * index1 + 7]));
	//			if (index2 < sizeOfRigidBodies)
	//			{
	//				b.position4d = vec4(positions4D[4 * index2], positions4D[4 * index2 + 1], positions4D[4 * index2 + 2], positions4D[4 * index2 + 3]);
	//				b.rotation4D = rotor4(rotations4D[8 * index2], bivec4(rotations4D[8 * index2 + 1], rotations4D[8 * index2 + 2], rotations4D[8 * index2 + 3], rotations4D[8 * index2 + 4], rotations4D[8 * index2 + 5], rotations4D[8 * index2 + 6]), quadvec4(rotations4D[8 * index2 + 7]));
	//			}
	//			vec4 contactPosAToWorld = body_pos_to_world(a, collisions[i].collisionManifold.contactPosA);
	//			vec4 contactPosBToWorld = body_pos_to_world(b, collisions[i].collisionManifold.contactPosB);
	//			vec4 aB = contactPosAToWorld - contactPosBToWorld;
	//			if (aB != vec4(0.f))
	//			{
	//				vec4 force4D = normalize(aB);
	//				vec4 velocity4DA = (a.mass > 0.f ? -force4D / a.mass : vec4(0.f));
	//				vec4 velocity4DB;
	//				if (index2 < sizeOfRigidBodies|| !isStatics[index2])velocity4DB = (b.mass > 0.f ? force4D / b.mass : vec4(0.f));
	//				else velocity4DB = vec4(0.f);
	//				//bivec4 angularVelocity4DA(wedge_v(collisions[i].collisionManifold.contactPosA - a.massCenter, rotate(a.rotation4D.reverse(), -force4D)));
	//				//bivec4 angularVelocity4DB(wedge_v(collisions[i].collisionManifold.contactPosB - b.massCenter, rotate(b.rotation4D.reverse(), force4D)));
	//				//float momentInertiaScalarA(get_moment_inertia_scalar(a, angularVelocity4DA));
	//				//float momentInertiaScalarB(get_moment_inertia_scalar(b, angularVelocity4DB));
	//				//angularVelocity4DA = inverse_moment_of_inertia(momentInertiaScalarA, angularVelocity4DA);
	//				//angularVelocity4DB = inverse_moment_of_inertia(momentInertiaScalarB, angularVelocity4DB);
	//				vec4 vatAtA = velocity4DA;// + body_vec_to_world(a, left_contract_bv(collisions[i].collisionManifold.contactPosA - a.massCenter, angularVelocity4DA));
	//				vec4 vatAtB = velocity4DB;// + body_vec_to_world(b, left_contract_bv(collisions[i].collisionManifold.contactPosB - b.massCenter, angularVelocity4DB));
	//				if (vatAtB != vatAtA)
	//				{
	//					float factor = collisions[i].collisionManifold.depth / length(vatAtB - vatAtA);
	//					collisions[i].collisionManifold.depth = length(aB);
	//					velocity4DA = velocity4DA * factor/4.f;
	//					velocity4DB = velocity4DB * factor/4.f;
	//					//angularVelocity4DA = angularVelocity4DA * factor;
	//					//angularVelocity4DB = angularVelocity4DB * factor;
	//					atomicAdd(&positions4D[4 * index1], velocity4DA.x), atomicAdd(&positions4D[4 * index1 + 1], velocity4DA.y), atomicAdd(&positions4D[4 * index1 + 2], velocity4DA.z), atomicAdd(&positions4D[4 * index1 + 3], velocity4DA.w);
	//					if (index2 < sizeOfRigidBodies)atomicAdd(&positions4D[4 * index2], velocity4DB.x), atomicAdd(&positions4D[4 * index2 + 1], velocity4DB.y), atomicAdd(&positions4D[4 * index2 + 2], velocity4DB.z), atomicAdd(&positions4D[4 * index2 + 3], velocity4DB.w);
	//					//atomicAdd(&angularVelocitys4D[6 * index1], angularVelocity4DA.xy), atomicAdd(&angularVelocitys4D[6 * index1 + 1], angularVelocity4DA.xz), atomicAdd(&angularVelocitys4D[6 * index1 + 2], angularVelocity4DA.xw), atomicAdd(&angularVelocitys4D[6 * index1 + 3], angularVelocity4DA.yz), atomicAdd(&angularVelocitys4D[6 * index1 + 4], angularVelocity4DA.yw), atomicAdd(&angularVelocitys4D[6 * index1 + 5], angularVelocity4DA.zw);
	//					//atomicAdd(&angularVelocitys4D[6 * index2], angularVelocity4DB.xy), atomicAdd(&angularVelocitys4D[6 * index2 + 1], angularVelocity4DB.xz), atomicAdd(&angularVelocitys4D[6 * index2 + 2], angularVelocity4DB.xw), atomicAdd(&angularVelocitys4D[6 * index2 + 3], angularVelocity4DB.yz), atomicAdd(&angularVelocitys4D[6 * index2 + 4], angularVelocity4DB.yw), atomicAdd(&angularVelocitys4D[6 * index2 + 5], angularVelocity4DB.zw);
	//
	//				}
	//			}
	//		}
	//	}
	//}

	//sizesOfCollisionsVec[index] = 0;
	//a.position4d = vec4(positions4D[4 * index1], positions4D[4 * index1 + 1], positions4D[4 * index1 + 2], positions4D[4 * index1 + 3]);
	//a.rotation4D = rotor4(rotations4D[8 * index1], bivec4(rotations4D[8 * index1 + 1], rotations4D[8 * index1 + 2], rotations4D[8 * index1 + 3], rotations4D[8 * index1 + 4], rotations4D[8 * index1 + 5], rotations4D[8 * index1 + 6]), quadvec4(rotations4D[8 * index1 + 7]));
	//if (index2 < sizeOfRigidBodies)
	//{
	//	b.position4d = vec4(positions4D[4 * index2], positions4D[4 * index2 + 1], positions4D[4 * index2 + 2], positions4D[4 * index2 + 3]);
	//	b.rotation4D = rotor4(rotations4D[8 * index2], bivec4(rotations4D[8 * index2 + 1], rotations4D[8 * index2 + 2], rotations4D[8 * index2 + 3], rotations4D[8 * index2 + 4], rotations4D[8 * index2 + 5], rotations4D[8 * index2 + 6]), quadvec4(rotations4D[8 * index2 + 7]));
	//}
	//if (index2 != -1)
	//{
	//	if (!isStatics[index1])
	//	{
	//		detect_rigidbody_or_wall(index1, index2, a, b, vertices4DsVec, sizesOfVertices4D, verticesStartIndices, collisions, &sizesOfCollisionsVec[index]);
	//	}
	//	else if (isStatics[index1] && !isStatics[index2] && index2 < sizeOfRigidBodies)
	//	{
	//		detect_rigidbody_or_wall(index2, index1, b, a, vertices4DsVec, sizesOfVertices4D, verticesStartIndices, collisions, &sizesOfCollisionsVec[index]);
	//	}
	//}
	//else
	//{
	//	if (!isStatics[index1])
	//	{
	//		detect_rigidbody_edge(index1, a, vertices4DsVec, sizesOfVertices4D, verticesStartIndices, collisions, &sizesOfCollisionsVec[index]);
	//	}
	//}

	if (sizesOfCollisionsVec[index] > 0)
	{
		for (int i(_d_max(index - 1, 0)); i < pSizeOfPairs; i++)
		{
			if (pSizesOfCollisionsVec[i] > 0)
			{
				//velocitys4D[4 * index1 + 1] = 1.f; velocitys4D[4 * index2 + 1] = 1.f;
				if (index1 == pPairsVec[2 * i] && index2 == pPairsVec[2 * i + 1])
				{
					//velocitys4D[4 * index1 + 1] = 1.f; velocitys4D[4 * index2 + 1] = 1.f;
					for (int j(0); j < pSizesOfCollisionsVec[i]; j++)
					{
						//int pIndexI = 2 * (j + 32 * i);
						int pIndexF = 50 * (j + 32 * i);
						vec4 pContactPosA = vec4(constraintsVecFloatArray[pIndexF + 16], constraintsVecFloatArray[pIndexF + 17], constraintsVecFloatArray[pIndexF + 18], constraintsVecFloatArray[pIndexF + 19]);
						vec4 pContactPosB = vec4(constraintsVecFloatArray[pIndexF + 20], constraintsVecFloatArray[pIndexF + 21], constraintsVecFloatArray[pIndexF + 22], constraintsVecFloatArray[pIndexF + 23]);
						for (int k(0); k < sizesOfCollisionsVec[index]; k++)
						{
							
							//int indexI = 3 * (k + 32 * index);
							int indexF = 15 * (k + 32 * index);
							vec4 contactPosA = collisions[k].collisionManifold.contactPosA;
							vec4 contactPosB = collisions[k].collisionManifold.contactPosB;
							if (length(contactPosA - pContactPosA) < 0.1f &&
								length(contactPosB - pContactPosB) < 0.1f)
							{
								if (collisions[k].type == 1)
									if (constraintsVecFloatArray[pIndexF + 48] < constraintsVecFloatArray[pIndexF + 49])
									{
										vec4 contactPosAToWorld;
										vec4 contactPosBToWorld;
										if (a.isStatic && !b.isStatic)
										{
											contactPosAToWorld = body_pos_to_world(b, pContactPosA);
											if (index2 == -1)contactPosBToWorld = pContactPosB;
											else contactPosBToWorld = body_pos_to_world(a, pContactPosB);
										}
										else
										{
											contactPosAToWorld = body_pos_to_world(a, pContactPosA);
											if (index2 == -1)contactPosBToWorld = pContactPosB;
											else contactPosBToWorld = body_pos_to_world(b, pContactPosB);
										}
										if (contactPosAToWorld != contactPosBToWorld)
										{
											vec4 normal2(contactPosAToWorld - contactPosBToWorld);
											vec4 normalE(normalize(normal2));
											collisions[k].collisionManifold.depth = length(normal2);
											collisions[k].collisionManifold.normal = normalE;
										}
										collisions[k].collisionManifold.contactPosA = pContactPosA;
										collisions[k].collisionManifold.contactPosB = pContactPosB;
									}
								vec4 impulse = vec4(constraintsVecFloatArray[pIndexF + 42], constraintsVecFloatArray[pIndexF + 43], constraintsVecFloatArray[pIndexF + 44], constraintsVecFloatArray[pIndexF + 45]);
								resolve_impulse(a, index1, velocitys4D, angularVelocitys4D, -impulse, pContactPosA, constraintsVecFloatArray[pIndexF + 46]);
								if (index2 < sizeOfRigidBodies && index2 != -1)
								{
									resolve_impulse(b, index2, velocitys4D, angularVelocitys4D, impulse, pContactPosB, constraintsVecFloatArray[pIndexF + 47]);
								}
							}
						}
					}
					break;
				}
			}
		}
	}
	if (sizesOfCollisionsVec[index] > 0)
	{
		if (index2 < sizeOfRigidBodies)
		{
			if (!a.isStatic && !b.isStatic)
			{
				if (!a.isContinueSleep || length(a.velocity4d) > 0.2f || length(a.angularVelocity4D) > 1.6f || !b.isContinueSleep || length(b.velocity4d) > 0.2f || length(b.angularVelocity4D) > 1.6f)
				{
					isSleeps[index1] = isSleeps[index2] = false, isContinueSleeps[index1] = isContinueSleeps[index2] = false;
				}
			}
		}
	}
	for (int i(0); i < sizesOfCollisionsVec[index]; i++)
	{
		//velocitys4D[4 * index1] = -collisions[i].collisionManifold.normal.x;// * vertices4DsVec[4*verticesStartIndices[a.primitiveId]];
		//velocitys4D[4 * index1 + 1] = -collisions[i].collisionManifold.normal.y;// * vertices4DsVec[4*verticesStartIndices[a.primitiveId]+1];
		//velocitys4D[4 * index1 + 2] = -collisions[i].collisionManifold.normal.z;// * vertices4DsVec[4*verticesStartIndices[a.primitiveId]+2];
		//velocitys4D[4 * index1 + 3] = -collisions[i].collisionManifold.normal.w;// * vertices4DsVec[4*verticesStartIndices[a.primitiveId]+3];
		//velocitys4D[4 * index2] = collisions[i].collisionManifold.normal.x;// * vertices4DsVec[4*verticesStartIndices[b.primitiveId]];
		//velocitys4D[4 * index2 + 1] = collisions[i].collisionManifold.normal.y;// * vertices4DsVec[4*verticesStartIndices[b.primitiveId]+1];
		//velocitys4D[4 * index2 + 2] = collisions[i].collisionManifold.normal.z;// * vertices4DsVec[4*verticesStartIndices[b.primitiveId]+2];
		//velocitys4D[4 * index2 + 3] = collisions[i].collisionManifold.normal.w;// * vertices4DsVec[4*verticesStartIndices[b.primitiveId]+3];
		int arrayIndexI = 3 * (i + 32 * index);
		collisionsVecIntArray[arrayIndexI] = collisions[i].index1;
		collisionsVecIntArray[arrayIndexI + 1] = collisions[i].index2;
		collisionsVecIntArray[arrayIndexI + 2] = collisions[i].type;
		int arrayIndexF = 15 * (i + 32 * index);
		collisionsVecFloatArray[arrayIndexF] = collisions[i].massAdjustmentA;
		collisionsVecFloatArray[arrayIndexF + 1] = collisions[i].massAdjustmentB;
		collisionsVecFloatArray[arrayIndexF + 2] = collisions[i].collisionManifold.normal.x;
		collisionsVecFloatArray[arrayIndexF + 3] = collisions[i].collisionManifold.normal.y;
		collisionsVecFloatArray[arrayIndexF + 4] = collisions[i].collisionManifold.normal.z;
		collisionsVecFloatArray[arrayIndexF + 5] = collisions[i].collisionManifold.normal.w;
		collisionsVecFloatArray[arrayIndexF + 6] = collisions[i].collisionManifold.depth;
		collisionsVecFloatArray[arrayIndexF + 7] = collisions[i].collisionManifold.contactPosA.x;
		collisionsVecFloatArray[arrayIndexF + 8] = collisions[i].collisionManifold.contactPosA.y;
		collisionsVecFloatArray[arrayIndexF + 9] = collisions[i].collisionManifold.contactPosA.z;
		collisionsVecFloatArray[arrayIndexF + 10] = collisions[i].collisionManifold.contactPosA.w;
		collisionsVecFloatArray[arrayIndexF + 11] = collisions[i].collisionManifold.contactPosB.x;
		collisionsVecFloatArray[arrayIndexF + 12] = collisions[i].collisionManifold.contactPosB.y;
		collisionsVecFloatArray[arrayIndexF + 13] = collisions[i].collisionManifold.contactPosB.z;
		collisionsVecFloatArray[arrayIndexF + 14] = collisions[i].collisionManifold.contactPosB.w;
	}
}

__global__ static void adjust_collisions_and_apply_forces(
	float* positions4D,
	float* rotations4D,
	float* scales4D,
	bool* isGravitys,
	bool* isStatics,
	bool* isSleeps,
	bool* isContinueSleeps,
	float* forceSystems4D,
	int* sizesOfForces,
	int* forceStartIndices,
	float* masss,
	float* momentInertiaScalars,
	float* restitutions,
	float* mus,
	float* massCenters,
	float* velocitys4D,
	float* angularVelocitys4D,
	int* colTypes,
	int* primitiveIds,
	int* pairsVec,
	int* sizesOfPairs,
	int* pairsStartIndices,
	int* collisionsVecIntArray,
	float* collisionsVecFloatArray,
	int* sizesOfCollisionsVec,
	bool isTimeToSleep,
	float dt,
	int sizeOfRigidBodies
)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	int massAdjustmentA = 0;
	for (int i(0); i < sizesOfPairs[index]; i++)
	{
		massAdjustmentA += sizesOfCollisionsVec[pairsStartIndices[index] + i];
	}
	for (int i(0); i < sizesOfPairs[index]; i++)
	{
		int collisionIndex = pairsStartIndices[index] + i;
		int arrayIndexF = 32 * 15 * collisionIndex;
		for (int j(0); j < sizesOfCollisionsVec[collisionIndex]; j++)
		{
			collisionsVecFloatArray[15 * j + arrayIndexF] = massAdjustmentA;
			collisionsVecFloatArray[15 * j + arrayIndexF + 1] += sizesOfCollisionsVec[collisionIndex];
		}
	}
	_d_RigidBody4D a = copy_data_to_rigid_body(index, true, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
	if (a.isSleep)
	{
		if (a.isStatic)
		{
			if (isTimeToSleep)
			{
				a.isSleep = false;
			}
		}
	}
	else
	{
		if (!a.isStatic)
		{
			if (length(a.velocity4d) > 0.1f || length(a.angularVelocity4D) > 0.8f)
			{
				a.isContinueSleep = false;
			}
			if (isTimeToSleep)
			{
				if (a.isContinueSleep)
				{
					a.isSleep = true;
				}
				a.isContinueSleep = true;
			}
		}
	}
	if (!a.isStatic)
	{
		update_rigidbody_velocity4D(&a, forceSystems4D, dt);
	}
	if (a.is_dynamic())
	{
		if (a.isGravity)resolve_impulse(&a, vec4(0.f, -9.80665f * a.mass * dt, 0.f, 0.f));
		move_velocity_data_from_rigid_body(index, a, velocitys4D, angularVelocitys4D);
	}
	isSleeps[index] = a.isSleep, isContinueSleeps[index] = a.isContinueSleep;
}

__global__ static void get_and_solve_constraint(
	float* positions4D,
	float* rotations4D,
	float* scales4D,
	bool* isGravitys,
	bool* isStatics,
	bool* isSleeps,
	bool* isContinueSleeps,
	int* sizesOfForces,
	int* forceStartIndices,
	float* masss,
	float* momentInertiaScalars,
	float* restitutions,
	float* mus,
	float* massCenters,
	float* velocitys4D,
	float* angularVelocitys4D,
	int* colTypes,
	int* primitiveIds,
	int* pairsVec,
	int* sizesOfPairs,
	int* pairsStartIndices,
	int* collisionsVecIntArray,
	float* collisionsVecFloatArray,
	int* sizesOfCollisionsVec,
	int* constraintsVecIntArray,
	float* constraintsVecFloatArray,
	int sizeOfRigidBodies
)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index%32<sizesOfCollisionsVec[index/32])
	{
		int arrayIndexF(15 * index);
	_d_Collision collision = { collisionsVecIntArray[3 * index], collisionsVecIntArray[3 * index + 1], collisionsVecFloatArray[arrayIndexF], collisionsVecFloatArray[arrayIndexF + 1],
		{vec4(collisionsVecFloatArray[arrayIndexF + 2],collisionsVecFloatArray[arrayIndexF + 3], collisionsVecFloatArray[arrayIndexF + 4], collisionsVecFloatArray[arrayIndexF + 5]),collisionsVecFloatArray[arrayIndexF + 6],
		vec4(collisionsVecFloatArray[arrayIndexF + 7], collisionsVecFloatArray[arrayIndexF + 8],collisionsVecFloatArray[arrayIndexF + 9], collisionsVecFloatArray[arrayIndexF + 10]),
	vec4(collisionsVecFloatArray[arrayIndexF + 11], collisionsVecFloatArray[arrayIndexF + 12],collisionsVecFloatArray[arrayIndexF + 13], collisionsVecFloatArray[arrayIndexF + 14])}, collisionsVecIntArray[3 * index + 2] };
		_d_Constraint constraint;
		int index1 = collision.index1;
		int index2 = collision.index2;
		_d_RigidBody4D a = copy_data_to_rigid_body(index1, true, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
		if (index2 != -1)
		{
			_d_RigidBody4D b = copy_data_to_rigid_body(index2, index2 < sizeOfRigidBodies, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
			if (!a.isStatic && !b.isStatic)
			{
				get_constraint(a, b, collision, &constraint);
				move_data_from_constraint(index, constraint, constraintsVecIntArray, constraintsVecFloatArray);
				if (!(a.isSleep && b.isSleep))
				{
					for (int i(0); i < 140; i++) { solve(constraintsVecFloatArray, 50 * index, a, index1, b, index2, velocitys4D, angularVelocitys4D); }
				}
			}
			else if (!a.isStatic && b.isStatic)
			{
				get_constraint_of_wall(a, b, collision, &constraint);
				move_data_from_constraint(index, constraint, constraintsVecIntArray, constraintsVecFloatArray);
				if (!a.isSleep)
				{
					for (int i(0); i < 140; i++) { solve(constraintsVecFloatArray, 50 * index, a, index1, velocitys4D, angularVelocitys4D); }
				}
			}
			else if (a.isStatic && !b.isStatic)
			{
				get_constraint_of_wall(b, a, collision, &constraint);
				move_data_from_constraint(index, constraint, constraintsVecIntArray, constraintsVecFloatArray);
				if (!b.isSleep)
				{
					for (int i(0); i < 140; i++) { solve(constraintsVecFloatArray, 50 * index, b, index2, velocitys4D, angularVelocitys4D); }
				}
			}
		}
		else
		{
			if (!a.isStatic) {
				get_constraint(a, collision, &constraint);
				move_data_from_constraint(index, constraint, constraintsVecIntArray, constraintsVecFloatArray);
				if (!a.isSleep)
				{
					for (int i(0); i < 140; i++) { solve(constraintsVecFloatArray, 50 * index, a, index1, velocitys4D, angularVelocitys4D); }
				}
			}
		}
		
		//_d_RigidBody4D a = copy_data_to_rigid_body(index1, true, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, forceSystems4D, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
	}
}

//__global__ static void solve_constraint(
//	float* positions4D,
//	float* rotations4D,
//	float* scales4D,
//	bool* isGravitys,
//	bool* isStatics,
//	bool* isSleeps,
//	bool* isContinueSleeps,
//	float* forceSystems4D,
//	int* sizesOfForces,
//	int* forceStartIndices,
//	float* masss,
//	float* momentInertiaScalars,
//	float* restitutions,
//	float* mus,
//	float* massCenters,
//	float* velocitys4D,
//	float* angularVelocitys4D,
//	int* colTypes,
//	int* primitiveIds,
//	int* pairsVec,
//	int* sizesOfPairs,
//	int* pairsStartIndices,
//	int* collisionsVecIntArray,
//	float* collisionsVecFloatArray,
//	int* sizesOfCollisionsVec,
//	int* constraintsVecIntArray,
//	float* constraintsVecFloatArray,
//	int sizeOfRigidBodies
//)
//{
//	int index = blockDim.x * blockIdx.x + threadIdx.x;
//	if (index % 32 < sizesOfCollisionsVec[(index - index % 32) / 32])
//	{
//		int index1 = collisionsVecIntArray[3 * index];
//		int index2 = collisionsVecIntArray[3 * index + 1];
//		_d_RigidBody4D a = copy_data_to_rigid_body(index1, true, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, forceSystems4D, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
//		if (index2 != -1)
//		{
//			_d_RigidBody4D b = copy_data_to_rigid_body(index2, index2 < sizeOfRigidBodies, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, forceSystems4D, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
//			if (!a.isStatic && !b.isStatic)
//			{
//				for (int i(0); i < 192; i++) { solve(constraintsVecFloatArray, 50 * index, a, index1, b, index2, velocitys4D, angularVelocitys4D); }
//			}
//			else if (!a.isStatic && b.isStatic)
//			{
//				for (int i(0); i < 192; i++) { solve(constraintsVecFloatArray, 50 * index, a, index1, velocitys4D, angularVelocitys4D); }
//			}
//			else if (a.isStatic && !b.isStatic)
//			{
//				for (int i(0); i < 192; i++) { solve(constraintsVecFloatArray, 50 * index, b, index2, velocitys4D, angularVelocitys4D); }
//			}
//		}
//		else
//		{
//			if (!a.isStatic) {
//				for (int i(0); i < 192; i++) { solve(constraintsVecFloatArray, 50 * index, a, index1, velocitys4D, angularVelocitys4D); }
//			}
//		}
//	}
//}

//__global__ static void apply_velocitys(
//	float* velocitys4D,
//	float* angularVelocitys4D,
//	int* pairsVec,
//	int* sizesOfPairs,
//	int* pairsStartIndices,
//	int* collisionsVecIntArray,
//	float* collisionsVecFloatArray,
//	int* sizesOfCollisionsVec,
//	int* constraintsVecIntArray,
//	float* constraintsVecFloatArray,
//	float* velocitys4DAdd,
//	float* angularVelocitys4DAdd,
//	int sizeOfRigidBodies
//)
//{
//	int index = blockDim.x * blockIdx.x + threadIdx.x;
//	for (int j(0); j < pairsStartIndices[index]+sizesOfPairs[index]; j++)
//	{
//		for (int i(0); i < sizesOfCollisionsVec[j]; i++)
//		{
//			if (pairsVec[2 * j] == index)
//			{
//				int indexV = 2 * (32 * j + i);
//				velocitys4D[4 * index] += velocitys4DAdd[4 * indexV], velocitys4D[4 * index + 1] += velocitys4DAdd[4 * indexV + 1], velocitys4D[4 * index + 2] += velocitys4DAdd[4 * indexV + 2], velocitys4D[4 * index + 3] += velocitys4DAdd[4 * indexV + 3];
//				angularVelocitys4D[6 * index] += angularVelocitys4DAdd[6 * indexV], angularVelocitys4D[6 * index + 1] += angularVelocitys4DAdd[6 * indexV + 1], angularVelocitys4D[6 * index + 2] += angularVelocitys4DAdd[6 * indexV + 2], angularVelocitys4D[6 * index + 3] += angularVelocitys4DAdd[6 * indexV + 3], angularVelocitys4D[6 * index + 4] += angularVelocitys4DAdd[6 * indexV + 4], angularVelocitys4D[6 * index + 5] += angularVelocitys4DAdd[6 * indexV + 5];
//			}
//			if (pairsVec[2 * j + 1] == index)
//			{
//				int indexV = 2 * (32 * j + i);
//				velocitys4D[4 * index] += velocitys4DAdd[4 * indexV + 4], velocitys4D[4 * index + 1] += velocitys4DAdd[4 * indexV + 5], velocitys4D[4 * index + 2] += velocitys4DAdd[4 * indexV + 6], velocitys4D[4 * index + 3] += velocitys4DAdd[4 * indexV + 7];
//				angularVelocitys4D[6 * index] += angularVelocitys4DAdd[6 * indexV + 6], angularVelocitys4D[6 * index + 1] += angularVelocitys4DAdd[6 * indexV + 7], angularVelocitys4D[6 * index + 2] += angularVelocitys4DAdd[6 * indexV + 8], angularVelocitys4D[6 * index + 3] += angularVelocitys4DAdd[6 * indexV + 9], angularVelocitys4D[6 * index + 4] += angularVelocitys4DAdd[6 * indexV + 10], angularVelocitys4D[6 * index + 5] += angularVelocitys4DAdd[6 * indexV + 11];
//			}
//		}
//	}
//}

__global__ static void update_positions4D(
	float* positions4D,
	float* rotations4D,
	float* scales4D,
	bool* isGravitys,
	bool* isStatics,
	bool* isSleeps,
	bool* isContinueSleeps,
	int* sizesOfForces,
	int* forceStartIndices,
	float* masss,
	float* momentInertiaScalars,
	float* restitutions,
	float* mus,
	float* massCenters,
	float* velocitys4D,
	float* angularVelocitys4D,
	int* colTypes,
	int* primitiveIds,
	float* mats4,
	float dt,
	int sizeOfRigidBodies
)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	_d_RigidBody4D a = copy_data_to_rigid_body(index, true, positions4D, rotations4D, scales4D, isGravitys, isStatics, isSleeps, isContinueSleeps, sizesOfForces, forceStartIndices, masss, momentInertiaScalars, restitutions, mus, massCenters, velocitys4D, angularVelocitys4D, colTypes, primitiveIds);
	if (a.is_dynamic())
	{
		if (a.massCenter == vec4(0.f))
		{
			a.position4d = a.position4d + a.velocity4d * dt;
			a.rotation4D = update(a.rotation4D, a.angularVelocity4D * dt);

		}
		else
		{
			vec4 bodyMassCenterToWorld(body_pos_to_world(a, a.massCenter));
			a.rotation4D = update(a.rotation4D, a.angularVelocity4D * dt);
			vec4 bodyMassCenterToVec(body_vec_to_world(a, a.massCenter));
			a.position4d = bodyMassCenterToWorld - bodyMassCenterToVec + a.velocity4d * dt;
		}
		positions4D[4 * index] = a.position4d.x, positions4D[4 * index + 1] = a.position4d.y, positions4D[4 * index + 2] = a.position4d.z, positions4D[4 * index + 3] = a.position4d.w;
		rotations4D[8 * index] = a.rotation4D.s, rotations4D[8 * index + 1] = a.rotation4D.b.xy, rotations4D[8 * index + 2] = a.rotation4D.b.xz, rotations4D[8 * index + 3] = a.rotation4D.b.xw, rotations4D[8 * index + 4] = a.rotation4D.b.yz, rotations4D[8 * index + 5] = a.rotation4D.b.yw, rotations4D[8 * index + 6] = a.rotation4D.b.zw, rotations4D[8 * index + 7] = a.rotation4D.q.xyzw;
		to_matrix(a.rotation4D, a.scale4D, mats4, index);
	}
}

static float* positions4D;
static float* rotations4D;
static float* scales4D;
static bool* isGravitys;
static bool* isStatics;
static bool* isSleeps;
static bool* isContinueSleeps;
static float* forceSystems4D;
static int* sizesOfForces;
static int* forceStartIndices;
static float* masss;
static float* momentInertiaScalars;
static float* restitutions;
static float* mus;
static float* massCenters;
static float* velocitys4D;
static float* angularVelocitys4D;
static float* vertices4DsVec;
static int* sizesOfVertices4D;
static int* verticesStartIndices;
static int* colTypes;
static int* primitiveIds;
static int* pairsVec;
static int* sizesOfPairs;
static int* pairsStartIndices;
//static int* collisionsVecIntArray;
static float* collisionsVecFloatArray;
static int* sizesOfCollisionsVec;
static float* mats4;

static float* _d_positions4D;
static float* _d_rotations4D;
static float* _d_scales4D;
static bool* _d_isGravitys;
static bool* _d_isStatics;
static bool* _d_isSleeps;
static bool* _d_isContinueSleeps;
static float* _d_forceSystems4D;
static int* _d_sizesOfForces;
static int* _d_forceStartIndices;
static float* _d_masss;
static float* _d_momentInertiaScalars;
static float* _d_restitutions;
static float* _d_mus;
static float* _d_massCenters;
static float* _d_velocitys4D;
static float* _d_angularVelocitys4D;
static float* _d_vertices4DsVec;
static int* _d_sizesOfVertices4D;
static int* _d_verticesStartIndices;
static int* _d_colTypes;
static int* _d_primitiveIds;
static int* _d_pairsVec;
static int* _d_sizesOfPairs;
static int* _d_pairsStartIndices;
static int* _d_collisionsVecIntArray;
static float* _d_collisionsVecFloatArray;
static int* _d_sizesOfCollisionsVec;
static int* _d_constraintsVecIntArray;
static float* _d_constraintsVecFloatArray;
static float* _d_mats4;
static int* _d_pPairsVec;
static int* _d_pSizesOfPairs;
static int* _d_pPairsStartIndices;
static int* _d_pSizesOfCollisionsVec;

static size_t pSizeOfPairs;

static void init_physics4d_function()
{
	const int sizeOfRigidBodies = 2500;
	const int sizeOfRigidBodiesAndWalls = 5000;
	size_t sizeList[38];
	sizeList[0] = 4 * sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[1] = 8 * sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[2] = 4 * sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[3] = sizeof(bool) * sizeOfRigidBodies;
	sizeList[4] = sizeof(bool) * sizeOfRigidBodies;
	sizeList[5] = sizeof(bool) * sizeOfRigidBodies;
	sizeList[6] = sizeof(bool) * sizeOfRigidBodies;
	int forceVecSize = 0; for (int i(0); i < sizeOfRigidBodies; i++) forceVecSize += 1;
	sizeList[7] = 8 * sizeof(float) * forceVecSize;
	sizeList[8] = sizeof(int) * sizeOfRigidBodies;
	sizeList[9] = sizeof(int) * sizeOfRigidBodies;
	sizeList[10] = sizeof(float) * sizeOfRigidBodies;
	sizeList[11] = sizeof(float) * sizeOfRigidBodies;
	sizeList[12] = sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[13] = sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[14] = 4 * sizeof(float) * sizeOfRigidBodies;
	sizeList[15] = 4 * sizeof(float) * sizeOfRigidBodies;
	sizeList[16] = 6 * sizeof(float) * sizeOfRigidBodies;
	sizeList[17] = sizeof(int) * sizeOfRigidBodiesAndWalls;
	int vertices4DVecSize = 0; for (int i(3); i < 13; i++) { vertices4DVecSize += get_vertices_size(i); }
	sizeList[18] = 4 * sizeof(float) * vertices4DVecSize;
	sizeList[19] = sizeof(int) * 13;
	sizeList[20] = sizeof(int) * 13;
	sizeList[24] = sizeof(int) * sizeOfRigidBodiesAndWalls;
	sizeList[25] = 16 * sizeof(int) * sizeOfRigidBodies;
	sizeList[26] = 8 * sizeof(int) * sizeOfRigidBodies;
	sizeList[27] = 8 * sizeof(int) * sizeOfRigidBodies;
	sizeList[28] = 32 * 8 * 3 * sizeof(int) * sizeOfRigidBodies;
	sizeList[29] = 32 * 8 * 15 * sizeof(float) * sizeOfRigidBodies;
	sizeList[30] = 8 * sizeof(int) * sizeOfRigidBodies;
	sizeList[31] = 32 * 8 * 2 * sizeof(int) * sizeOfRigidBodies;
	sizeList[32] = 32 * 8 * 50 * sizeof(float) * sizeOfRigidBodies;
	sizeList[33] = 16 * sizeof(float) * sizeOfRigidBodies;
	sizeList[34] = 16 * sizeof(int) * sizeOfRigidBodies;
	sizeList[35] = 8 * sizeof(int) * sizeOfRigidBodies;
	sizeList[36] = 8 * sizeof(int) * sizeOfRigidBodies;
	sizeList[37] = 8 * sizeof(int) * sizeOfRigidBodies;
	
	hipHostMalloc((void**)&positions4D, sizeList[0]);
	hipHostMalloc((void**)&rotations4D, sizeList[1]);
	hipHostMalloc((void**)&scales4D, sizeList[2]);
	hipHostMalloc((void**)&isGravitys, sizeList[3]);
	hipHostMalloc((void**)&isStatics, sizeList[4]);
	hipHostMalloc((void**)&isSleeps, sizeList[5]);
	hipHostMalloc((void**)&isContinueSleeps, sizeList[6]);
	hipHostMalloc((void**)&forceSystems4D, sizeList[7]);
	hipHostMalloc((void**)&sizesOfForces, sizeList[8]);
	hipHostMalloc((void**)&forceStartIndices, sizeList[9]);
	hipHostMalloc((void**)&masss, sizeList[10]);
	hipHostMalloc((void**)&momentInertiaScalars, sizeList[11]);
	hipHostMalloc((void**)&restitutions, sizeList[12]);
	hipHostMalloc((void**)&mus, sizeList[13]);
	hipHostMalloc((void**)&massCenters, sizeList[14]);
	hipHostMalloc((void**)&velocitys4D, sizeList[15]);
	hipHostMalloc((void**)&angularVelocitys4D, sizeList[16]);
	hipHostMalloc((void**)&colTypes, sizeList[17]);
	hipHostMalloc((void**)&vertices4DsVec, sizeList[18]);
	hipHostMalloc((void**)&sizesOfVertices4D, sizeList[19]);
	hipHostMalloc((void**)&verticesStartIndices, sizeList[20]);
	hipHostMalloc((void**)&primitiveIds, sizeList[24]);
	hipHostMalloc((void**)&pairsVec, sizeList[25]);
	hipHostMalloc((void**)&sizesOfPairs, sizeList[26]);
	hipHostMalloc((void**)&pairsStartIndices, sizeList[27]);
	//hipHostMalloc((void**)&collisionsVecIntArray, sizeList[28]);
	hipHostMalloc((void**)&collisionsVecFloatArray, sizeList[29]);
	hipHostMalloc((void**)&sizesOfCollisionsVec, sizeList[30]);
	hipHostMalloc((void**)&mats4, sizeList[33]);

	hipMalloc((void**)&_d_positions4D, sizeList[0]);
	hipMalloc((void**)&_d_rotations4D, sizeList[1]);
	hipMalloc((void**)&_d_scales4D, sizeList[2]);
	hipMalloc((void**)&_d_isGravitys, sizeList[3]);
	hipMalloc((void**)&_d_isStatics, sizeList[4]);
	hipMalloc((void**)&_d_isSleeps, sizeList[5]);
	hipMalloc((void**)&_d_isContinueSleeps, sizeList[6]);
	hipMalloc((void**)&_d_forceSystems4D, sizeList[7]);
	hipMalloc((void**)&_d_sizesOfForces, sizeList[8]);
	hipMalloc((void**)&_d_forceStartIndices, sizeList[9]);
	hipMalloc((void**)&_d_masss, sizeList[10]);
	hipMalloc((void**)&_d_momentInertiaScalars, sizeList[11]);
	hipMalloc((void**)&_d_restitutions, sizeList[12]);
	hipMalloc((void**)&_d_mus, sizeList[13]);
	hipMalloc((void**)&_d_massCenters, sizeList[14]);
	hipMalloc((void**)&_d_velocitys4D, sizeList[15]);
	hipMalloc((void**)&_d_angularVelocitys4D, sizeList[16]);
	hipMalloc((void**)&_d_colTypes, sizeList[17]);
	hipMalloc((void**)&_d_vertices4DsVec, sizeList[18]);
	hipMalloc((void**)&_d_sizesOfVertices4D, sizeList[19]);
	hipMalloc((void**)&_d_verticesStartIndices, sizeList[20]);
	hipMalloc((void**)&_d_primitiveIds, sizeList[24]);
	hipMalloc((void**)&_d_pairsVec, sizeList[25]);
	hipMalloc((void**)&_d_sizesOfPairs, sizeList[26]);
	hipMalloc((void**)&_d_pairsStartIndices, sizeList[27]);
	hipMalloc((void**)&_d_collisionsVecIntArray, sizeList[28]);
	hipMalloc((void**)&_d_collisionsVecFloatArray, sizeList[29]);
	hipMalloc((void**)&_d_sizesOfCollisionsVec, sizeList[30]);
	hipMalloc((void**)&_d_constraintsVecIntArray, sizeList[31]);
	hipMalloc((void**)&_d_constraintsVecFloatArray, sizeList[32]);
	hipMalloc((void**)&_d_mats4, sizeList[33]);
	hipMalloc((void**)&_d_pPairsVec, sizeList[34]);
	hipMalloc((void**)&_d_pSizesOfPairs, sizeList[35]);
	hipMalloc((void**)&_d_pPairsStartIndices, sizeList[36]);
	hipMalloc((void**)&_d_pSizesOfCollisionsVec, sizeList[37]);

	int vertexStartIndex = 0;
	for (int i(3); i < 13; i++)
	{
		//if (i != 5)
		{
			sizesOfVertices4D[i] = get_vertices_size(i);
			verticesStartIndices[i] = vertexStartIndex;
			vertexStartIndex += sizesOfVertices4D[i];
			for (int j(0); j < sizesOfVertices4D[i]; j++) {
				int vIndex = 4 * (verticesStartIndices[i] + j);
				glm::vec4 vertex(get_vertex(i,j));
				vertices4DsVec[vIndex] = vertex.x, vertices4DsVec[vIndex + 1] = vertex.y, vertices4DsVec[vIndex + 2] = vertex.z, vertices4DsVec[vIndex + 3] = vertex.w;
			}
		}
	}
	sizesOfVertices4D[0] = 0;
	sizesOfVertices4D[1] = 0;
	sizesOfVertices4D[2] = 0;
	//sizesOfVertices4D[5] = 0;
	verticesStartIndices[0] = 0;
	verticesStartIndices[1] = 0;
	verticesStartIndices[2] = 0;
	//verticesStartIndices[5] = verticesStartIndices[4];
	hipMemcpy(_d_vertices4DsVec, vertices4DsVec, sizeList[18], hipMemcpyHostToDevice);
	hipMemcpy(_d_sizesOfVertices4D, sizesOfVertices4D, sizeList[19], hipMemcpyHostToDevice);
	hipMemcpy(_d_verticesStartIndices, verticesStartIndices, sizeList[20], hipMemcpyHostToDevice);
}

static void destroy_physics4d_function()
{
	hipHostFree(positions4D);
	hipHostFree(rotations4D);
	hipHostFree(scales4D);
	hipHostFree(isGravitys);
	hipHostFree(isStatics);
	hipHostFree(isSleeps);
	hipHostFree(isContinueSleeps);
	hipHostFree(forceSystems4D);
	hipHostFree(sizesOfForces);
	hipHostFree(forceStartIndices);
	hipHostFree(masss);
	hipHostFree(momentInertiaScalars);
	hipHostFree(restitutions);
	hipHostFree(mus);
	hipHostFree(massCenters);
	hipHostFree(velocitys4D);
	hipHostFree(angularVelocitys4D);
	hipHostFree(colTypes);
	hipHostFree(vertices4DsVec);
	hipHostFree(sizesOfVertices4D);
	hipHostFree(verticesStartIndices);
	hipHostFree(primitiveIds);
	hipHostFree(pairsVec);
	hipHostFree(sizesOfPairs);
	hipHostFree(pairsStartIndices);
	//hipHostFree(collisionsVecIntArray);
	hipHostFree(collisionsVecFloatArray);
	hipHostFree(sizesOfCollisionsVec);
	hipHostFree(mats4);

	hipFree(_d_positions4D);
	hipFree(_d_rotations4D);
	hipFree(_d_scales4D);
	hipFree(_d_isGravitys);
	hipFree(_d_isStatics);
	hipFree(_d_isSleeps);
	hipFree(_d_isContinueSleeps);
	hipFree(_d_forceSystems4D);
	hipFree(_d_sizesOfForces);
	hipFree(_d_forceStartIndices);
	hipFree(_d_masss);
	hipFree(_d_momentInertiaScalars);
	hipFree(_d_restitutions);
	hipFree(_d_mus);
	hipFree(_d_massCenters);
	hipFree(_d_velocitys4D);
	hipFree(_d_angularVelocitys4D);
	hipFree(_d_colTypes);
	hipFree(_d_vertices4DsVec);
	hipFree(_d_sizesOfVertices4D);
	hipFree(_d_verticesStartIndices);
	hipFree(_d_primitiveIds);
	hipFree(_d_pairsVec);
	hipFree(_d_sizesOfPairs);
	hipFree(_d_pairsStartIndices);
	hipFree(_d_collisionsVecIntArray);
	hipFree(_d_collisionsVecFloatArray);
	hipFree(_d_sizesOfCollisionsVec);
	hipFree(_d_constraintsVecIntArray);
	hipFree(_d_constraintsVecFloatArray);
	hipFree(_d_mats4);
	hipFree(_d_pPairsVec);
	hipFree(_d_pSizesOfPairs);
	hipFree(_d_pPairsStartIndices);
	hipFree(_d_pSizesOfCollisionsVec);
}

static float timeToSleep = 0.f;

static void update_cuda(
	size_t sizeOfPairs,
	size_t* sizeList,
	size_t* sizeList2,
	bool isTimeToSleep,
	float dt,
	int sizeOfRigidBodies)
{

	hipMemcpy(_d_positions4D, positions4D, sizeList[0], hipMemcpyHostToDevice);
	hipMemcpy(_d_rotations4D, rotations4D, sizeList[1], hipMemcpyHostToDevice);
	hipMemcpy(_d_scales4D, scales4D, sizeList[2], hipMemcpyHostToDevice);
	hipMemcpy(_d_isGravitys, isGravitys, sizeList[3], hipMemcpyHostToDevice);
	hipMemcpy(_d_isStatics, isStatics, sizeList[4], hipMemcpyHostToDevice);
	hipMemcpy(_d_isSleeps, isSleeps, sizeList[5], hipMemcpyHostToDevice);
	hipMemcpy(_d_isContinueSleeps, isContinueSleeps, sizeList[6], hipMemcpyHostToDevice);
	hipMemcpy(_d_forceSystems4D, forceSystems4D, sizeList[7], hipMemcpyHostToDevice);
	hipMemcpy(_d_sizesOfForces, sizesOfForces, sizeList[8], hipMemcpyHostToDevice);
	hipMemcpy(_d_forceStartIndices, forceStartIndices, sizeList[9], hipMemcpyHostToDevice);
	hipMemcpy(_d_masss, masss, sizeList[10], hipMemcpyHostToDevice);
	hipMemcpy(_d_momentInertiaScalars, momentInertiaScalars, sizeList[11], hipMemcpyHostToDevice);
	hipMemcpy(_d_restitutions, restitutions, sizeList[12], hipMemcpyHostToDevice);
	hipMemcpy(_d_mus, mus, sizeList[13], hipMemcpyHostToDevice);
	hipMemcpy(_d_massCenters, massCenters, sizeList[14], hipMemcpyHostToDevice);
	hipMemcpy(_d_velocitys4D, velocitys4D, sizeList[15], hipMemcpyHostToDevice);
	hipMemcpy(_d_angularVelocitys4D, angularVelocitys4D, sizeList[16], hipMemcpyHostToDevice);
	hipMemcpy(_d_colTypes, colTypes, sizeList[17], hipMemcpyHostToDevice);
	hipMemcpy(_d_primitiveIds, primitiveIds, sizeList[24], hipMemcpyHostToDevice);
	hipMemcpy(_d_pairsVec, pairsVec, sizeList[25], hipMemcpyHostToDevice);
	hipMemcpy(_d_sizesOfPairs, sizesOfPairs, sizeList[26], hipMemcpyHostToDevice);
	hipMemcpy(_d_pairsStartIndices, pairsStartIndices, sizeList[27], hipMemcpyHostToDevice);

	detect_collision << <sizeOfPairs, 1 >> > (_d_positions4D, _d_rotations4D, _d_scales4D, _d_isGravitys, _d_isStatics, _d_isSleeps, _d_isContinueSleeps, _d_sizesOfForces, _d_forceStartIndices, _d_masss, _d_momentInertiaScalars, _d_restitutions, _d_mus, _d_massCenters, _d_velocitys4D, _d_angularVelocitys4D, _d_colTypes, _d_vertices4DsVec, _d_sizesOfVertices4D, _d_verticesStartIndices, _d_primitiveIds, _d_pairsVec, _d_sizesOfPairs, _d_pairsStartIndices, _d_collisionsVecIntArray, _d_collisionsVecFloatArray, _d_sizesOfCollisionsVec, _d_constraintsVecFloatArray, _d_pPairsVec, _d_pSizesOfPairs, _d_pPairsStartIndices, _d_pSizesOfCollisionsVec, pSizeOfPairs, dt, sizeOfRigidBodies);

	hipDeviceSynchronize();

	adjust_collisions_and_apply_forces << <sizeOfRigidBodies, 1 >> > (_d_positions4D, _d_rotations4D, _d_scales4D, _d_isGravitys, _d_isStatics, _d_isSleeps, _d_isContinueSleeps, _d_forceSystems4D, _d_sizesOfForces, _d_forceStartIndices, _d_masss, _d_momentInertiaScalars, _d_restitutions, _d_mus, _d_massCenters, _d_velocitys4D, _d_angularVelocitys4D, _d_colTypes, primitiveIds, _d_pairsVec, _d_sizesOfPairs, _d_pairsStartIndices, _d_collisionsVecIntArray, _d_collisionsVecFloatArray, _d_sizesOfCollisionsVec, isTimeToSleep, dt, sizeOfRigidBodies);

	hipDeviceSynchronize();

	get_and_solve_constraint << <sizeOfPairs, 32 >> > (_d_positions4D, _d_rotations4D, _d_scales4D, _d_isGravitys, _d_isStatics, _d_isSleeps, _d_isContinueSleeps, _d_sizesOfForces, _d_forceStartIndices, _d_masss, _d_momentInertiaScalars, _d_restitutions, _d_mus, _d_massCenters, _d_velocitys4D, _d_angularVelocitys4D, _d_colTypes, _d_primitiveIds, _d_pairsVec, _d_sizesOfPairs, _d_pairsStartIndices, _d_collisionsVecIntArray, _d_collisionsVecFloatArray, _d_sizesOfCollisionsVec, _d_constraintsVecIntArray, _d_constraintsVecFloatArray, sizeOfRigidBodies);

	hipDeviceSynchronize();

	update_positions4D << <sizeOfRigidBodies, 1 >> > (_d_positions4D, _d_rotations4D, _d_scales4D, _d_isGravitys, _d_isStatics, _d_isSleeps, _d_isContinueSleeps, _d_sizesOfForces, _d_forceStartIndices, _d_masss, _d_momentInertiaScalars, _d_restitutions, _d_mus, _d_massCenters, _d_velocitys4D, _d_angularVelocitys4D, _d_colTypes, _d_primitiveIds, _d_mats4, dt, sizeOfRigidBodies);

	hipDeviceSynchronize();

	hipMemcpy(_d_pPairsVec, _d_pairsVec, sizeList[25], hipMemcpyDeviceToDevice);
	hipMemcpy(_d_pSizesOfPairs, _d_sizesOfPairs, sizeList[26], hipMemcpyDeviceToDevice);
	hipMemcpy(_d_pPairsStartIndices, _d_pairsStartIndices, sizeList[27], hipMemcpyDeviceToDevice);
	hipMemcpy(_d_pSizesOfCollisionsVec, _d_sizesOfCollisionsVec, sizeList[25], hipMemcpyDeviceToDevice);
	
	pSizeOfPairs = sizeOfPairs;

	hipMemcpy(positions4D, _d_positions4D, sizeList2[0], hipMemcpyDeviceToHost);
	hipMemcpy(rotations4D, _d_rotations4D, sizeList2[1], hipMemcpyDeviceToHost);
	hipMemcpy(isSleeps, _d_isSleeps, sizeList[5], hipMemcpyDeviceToHost);
	hipMemcpy(isContinueSleeps, _d_isContinueSleeps, sizeList[6], hipMemcpyDeviceToHost);
	hipMemcpy(velocitys4D, _d_velocitys4D, sizeList[15], hipMemcpyDeviceToHost);
	hipMemcpy(angularVelocitys4D, _d_angularVelocitys4D, sizeList[16], hipMemcpyDeviceToHost);
	//hipMemcpy(collisionsVecIntArray, _d_collisionsVecIntArray, 32 * 3 * sizesOfPairs[0], hipMemcpyDeviceToHost);
	hipMemcpy(collisionsVecFloatArray, _d_collisionsVecFloatArray, 32 * 15 * sizesOfPairs[0] * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(sizesOfCollisionsVec, _d_sizesOfCollisionsVec, sizesOfPairs[0] * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(mats4, _d_mats4, sizeList[28], hipMemcpyDeviceToHost);
}

static void phy_function(std::vector<RigidBody4D*> rigidBodies4d, std::vector<Wall4D*> walls4d, std::vector < std::vector<Collision>>* collisionsVec)
{
	size_t sizeList[29];
	size_t sizeList2[2];
	int sizeOfRigidBodies = rigidBodies4d.size();
	int sizeOfRigidBodiesAndWalls = size_of_objects4d(rigidBodies4d, walls4d);
	sizeList[0] = 4 * sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[1] = 8 * sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[2] = 4 * sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[3] = sizeof(bool) * sizeOfRigidBodies;
	sizeList[4] = sizeof(bool) * sizeOfRigidBodies;
	sizeList[5] = sizeof(bool) * sizeOfRigidBodies;
	sizeList[6] = sizeof(bool) * sizeOfRigidBodies;
	int forceVecSize = 0; for (int i(0); i < sizeOfRigidBodies; i++) forceVecSize += rigidBodies4d[i]->forceSystems4D.size();
	sizeList[7] = 8 * sizeof(float) * forceVecSize;
	sizeList[8] = sizeof(int) * sizeOfRigidBodies;
	sizeList[9] = sizeof(int) * sizeOfRigidBodies;
	sizeList[10] = sizeof(float) * sizeOfRigidBodies;
	sizeList[11] = sizeof(float) * sizeOfRigidBodies;
	sizeList[12] = sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[13] = sizeof(float) * sizeOfRigidBodiesAndWalls;
	sizeList[14] = 4 * sizeof(float) * sizeOfRigidBodies;
	sizeList[15] = 4 * sizeof(float) * sizeOfRigidBodies;
	sizeList[16] = 6 * sizeof(float) * sizeOfRigidBodies;
	sizeList[17] = sizeof(int) * sizeOfRigidBodiesAndWalls;
	sizeList[24] = sizeof(int) * sizeOfRigidBodiesAndWalls;
	sizeList[28] = 16 * sizeof(float) * sizeOfRigidBodies;

	sizeList2[0] = 4 * sizeof(float) * sizeOfRigidBodies;
	sizeList2[1] = 8 * sizeof(float) * sizeOfRigidBodies;

	int forcesStartIndex = 0;
	int pairsStartIndex = 0;
	for (int i(0); i < sizeOfRigidBodies; i++)
	{
		RigidBody4D* a(rigidBodies4d[i]);
		isGravitys[i] = a->isGravity;
		isStatics[i] = a->is_static();
		isSleeps[i] = a->isSleep;
		isContinueSleeps[i] = a->isContinueSleep;

		sizesOfForces[i] = a->forceSystems4D.size();
		forceStartIndices[i] = forcesStartIndex;
		for (int j(0); j < a->forceSystems4D.size(); j++){
			//std::cout << forceStartIndices[i] << '\n';
			int fIndex = 8 * (forcesStartIndex + j);
			forceSystems4D[fIndex] = a->forceSystems4D[j]->force4D.x, forceSystems4D[fIndex + 1] = a->forceSystems4D[j]->force4D.y, forceSystems4D[fIndex + 2] = a->forceSystems4D[j]->force4D.z, forceSystems4D[fIndex + 3] = a->forceSystems4D[j]->force4D.w;
			forceSystems4D[fIndex + 4] = a->forceSystems4D[j]->position4d.x, forceSystems4D[fIndex + 5] = a->forceSystems4D[j]->position4d.y, forceSystems4D[fIndex + 6] = a->forceSystems4D[j]->position4d.z, forceSystems4D[fIndex + 7] = a->forceSystems4D[j]->position4d.w;
		}
		forcesStartIndex += a->forceSystems4D.size();

		masss[i] = a->mass;
		momentInertiaScalars[i] = a->momentInertiaScalar;
		massCenters[4 * i] = a->massCenter.x, massCenters[4 * i + 1] = a->massCenter.y, massCenters[4 * i + 2] = a->massCenter.z, massCenters[4 * i + 3] = a->massCenter.w;
		velocitys4D[4 * i] = a->velocity4d.x, velocitys4D[4 * i + 1] = a->velocity4d.y, velocitys4D[4 * i + 2] = a->velocity4d.z, velocitys4D[4 * i + 3] = a->velocity4d.w;
		angularVelocitys4D[6 * i] = a->angularVelocity4D.xy, angularVelocitys4D[6 * i + 1] = a->angularVelocity4D.xz, angularVelocitys4D[6 * i + 2] = a->angularVelocity4D.xw, angularVelocitys4D[6 * i + 3] = a->angularVelocity4D.yz, angularVelocitys4D[6 * i + 4] = a->angularVelocity4D.yw, angularVelocitys4D[6 * i + 5] = a->angularVelocity4D.zw;

		sizesOfPairs[i] = a->rigidBodyIndices.size() + a->wallIndices.size() + 1;
		pairsStartIndices[i] = pairsStartIndex;
		for (size_t j(0); j < a->rigidBodyIndices.size(); j++)
		{
			int pIndex = 2 * (pairsStartIndex + j);
			pairsVec[pIndex] = i, pairsVec[pIndex+1] = a->rigidBodyIndices[j];
		}
		for (size_t j(0); j < a->wallIndices.size(); j++)
		{
			int pIndex = 2 * (pairsStartIndex + a->rigidBodyIndices.size() + j);
			pairsVec[pIndex] = i, pairsVec[pIndex+1] = a->wallIndices[j] + sizeOfRigidBodies;
		}
		int pIndex = 2 * (pairsStartIndex + a->rigidBodyIndices.size() + a->wallIndices.size());
		pairsVec[pIndex] = i, pairsVec[pIndex + 1] = -1;
		pairsStartIndex += a->rigidBodyIndices.size() + a->wallIndices.size() + 1;
	}
	for (int i(0); i < sizeOfRigidBodiesAndWalls; i++)
	{
		Object4D* a(find_object4d(rigidBodies4d, walls4d, i));
		positions4D[4 * i] = a->position4d.x, positions4D[4 * i + 1] = a->position4d.y, positions4D[4 * i + 2] = a->position4d.z, positions4D[4 * i + 3] = a->position4d.w;
		rotations4D[8 * i] = a->rotation4D.s, rotations4D[8 * i + 1] = a->rotation4D.b.xy, rotations4D[8 * i + 2] = a->rotation4D.b.xz, rotations4D[8 * i + 3] = a->rotation4D.b.xw, rotations4D[8 * i + 4] = a->rotation4D.b.yz, rotations4D[8 * i + 5] = a->rotation4D.b.yw, rotations4D[8 * i + 6] = a->rotation4D.b.zw, rotations4D[8 * i + 7] = a->rotation4D.q.xyzw;
		scales4D[4 * i] = a->scale4D.x, scales4D[4 * i + 1] = a->scale4D.y, scales4D[4 * i + 2] = a->scale4D.z, scales4D[4 * i + 3] = a->scale4D.w;
		restitutions[i] = a->restitution;
		mus[i] = a->mu;
		colTypes[i] = a->colType;
		primitiveIds[i] = a->primitive4D.id;
	}

	sizeList[25] = 2 * sizeof(int) * pairsStartIndex;
	sizeList[26] = sizeof(int) * pairsStartIndex;
	sizeList[27] = sizeof(int) * pairsStartIndex;

	const float dt = 1 / 60.f;
	bool isTimeToSleep(false);
	timeToSleep += dt;
	if (timeToSleep > 2.f)
	{
		isTimeToSleep = true;
		timeToSleep = 0.f;
	}

	update_cuda(pairsStartIndex, sizeList, sizeList2, isTimeToSleep, dt, sizeOfRigidBodies);

	for (int i(0); i < sizeOfRigidBodies; i++)
	{
		RigidBody4D* a(rigidBodies4d[i]);
		if (!isSleeps[i] && !isStatics[i])
		{
			a->set_position4D(glm::vec4(positions4D[4 * i], positions4D[4 * i + 1], positions4D[4 * i + 2], positions4D[4 * i + 3]));
			a->set_rotation4D(alg::rotor4(rotations4D[8 * i], alg::bivec4(rotations4D[8 * i + 1], rotations4D[8 * i + 2], rotations4D[8 * i + 3], rotations4D[8 * i + 4], rotations4D[8 * i + 5], rotations4D[8 * i + 6]), alg::quadvec4(rotations4D[8 * i + 7]))
				, glm::mat4(mats4[16 * i], mats4[16 * i + 1], mats4[16 * i + 2], mats4[16 * i + 3], mats4[16 * i + 4], mats4[16 * i + 5], mats4[16 * i + 6], mats4[16 * i + 7], mats4[16 * i + 8], mats4[16 * i + 9], mats4[16 * i + 10], mats4[16 * i + 11], mats4[16 * i + 12], mats4[16 * i + 13], mats4[16 * i + 14], mats4[16 * i + 15]));
		}
		a->isSleep = isSleeps[i];
		a->isContinueSleep = isContinueSleeps[i];
		for (size_t j(0); j < a->forceSystems4D.size(); j++)a->forceSystems4D[j] = nullptr, free(a->forceSystems4D[j]); a->forceSystems4D.clear();
		a->velocity4d = glm::vec4(velocitys4D[4 * i], velocitys4D[4 * i + 1], velocitys4D[4 * i + 2], velocitys4D[4 * i + 3]);
		a->angularVelocity4D = alg::bivec4(angularVelocitys4D[6 * i], angularVelocitys4D[6 * i + 1], angularVelocitys4D[6 * i + 2], angularVelocitys4D[6 * i + 3], angularVelocitys4D[6 * i + 4], angularVelocitys4D[6 * i + 5]);
	}
	for (size_t i(0); i < collisionsVec->size(); i++)
	{
		collisionsVec[i].clear();
	}
	collisionsVec->clear();
	for (int j(0); j < sizesOfPairs[0]; j++)
	{
		//std::cout << "j=" << j << "collisionSize=" << sizesOfCollisionsVec[j] << '\n';
		if (sizesOfCollisionsVec[j] > 0)
		{
			std::vector<Collision> collisions;
			for (int i(0); i < sizesOfCollisionsVec[j]; i++)
			{
				size_t indexI = 3 * (32 * j + i);
				size_t indexF = 15 * (32 * j + i);
				collisions.push_back(Collision(pairsVec[2 * j], pairsVec[2 * j + 1], 0.f, 0.f, glm::vec4(collisionsVecFloatArray[indexF + 2], collisionsVecFloatArray[indexF + 3], collisionsVecFloatArray[indexF + 4], collisionsVecFloatArray[indexF + 5]), 0.f, glm::vec4(0.f), glm::vec4(0.f)));
			}
			collisionsVec->push_back(collisions);
		}
	}
}