#include "hip/hip_runtime.h"
#pragma once
#include "algebra.cu"

//__device__ static struct _d_ForceSystem4D
//{
//	vec4 force4D;
//	vec4 position4d;
//	//explicit _d_ForceSystem4D(vec4 force4D, vec4 position4d)
//	//{
//	//	this->force4D = force4D;
//	//	this->position4d = position4d;
//	//};
//};

__device__ static struct _d_RigidBody4D
{
	vec4 position4d;
	rotor4 rotation4D;
	vec4 scale4D;
	bool isGravity;
	bool isStatic;
	bool isSleep;
	bool isContinueSleep;
	int sizeOfForces;
	int forcesStartIndex;
	float mass;
	float momentInertiaScalar;
	float restitution;
	float mu;
	vec4 massCenter;
	vec4 velocity4d;
	bivec4 angularVelocity4D;
	int colType;
	int primitiveId;
	__device__ explicit _d_RigidBody4D(
		vec4 position4d,
		rotor4 rotation4D,
		vec4 scale4D,
		bool isGravity,
		bool isStatic,
		bool isSleep,
		bool isContinueSleep,
		int sizeOfForces,
		int forcesStartIndex,
		float mass,
		float momentInertiaScalar,
		float restitution,
		float mu,
		vec4 massCenter,
		vec4 velocity4d,
		bivec4 angularVelocity4D,
		int colType,
	    int primitiveId
		)
	{
		this->position4d = position4d;
		this->rotation4D = rotation4D;
		this->scale4D = scale4D;
		this->isGravity = isGravity;
		this->isStatic = isStatic;
		this->isSleep = isSleep;
		this->isContinueSleep = isContinueSleep;
		this->sizeOfForces = sizeOfForces;
		this->forcesStartIndex = forcesStartIndex;
		this->mass = mass;
		this->momentInertiaScalar = momentInertiaScalar;
		this->restitution = restitution;
		this->mu = mu;
		this->massCenter = massCenter;
		this->velocity4d = velocity4d;
		this->angularVelocity4D = angularVelocity4D;
		this->colType = colType;
		this->primitiveId = primitiveId;
	}
	__device__ explicit _d_RigidBody4D(
		vec4 position4d,
		rotor4 rotation4D,
		vec4 scale4D,
		float restitution,
		float mu,
		int colType,
		int primitiveId
	)
	{
		this->position4d = position4d;
		this->rotation4D = rotation4D;
		this->scale4D = scale4D;
		this->isStatic = true;
		this->restitution = restitution;
		this->mu = mu;
		this->colType = colType;
		this->primitiveId = primitiveId;
	}
	__device__ bool is_dynamic()
	{
		return !(this->isStatic || this->isSleep);
	}
};

__device__ static struct _d_HingeConstraint4D
{
	vec4 worldPosition4D;
	vec4 bodyPosition4D1;
	vec4 bodyPosition4D2;
	int index1;
	int index2;
	vec4 normal;
	float depth;
	__device__ explicit _d_HingeConstraint4D(
		vec4 worldPosition4D,
		vec4 bodyPosition4D1,
		vec4 bodyPosition4D2,
		int index1,
		int index2,
		vec4 normal,
		float depth
	)
	{
		this->worldPosition4D = worldPosition4D;
		this->bodyPosition4D1 = bodyPosition4D1;
		this->bodyPosition4D2 = bodyPosition4D2;
		this->index1 = index1;
		this->index2 = index2;
		this->normal = normal;
		this->depth = depth;
	}
};


__device__ static vec4 body_vec_to_world(rotor4 r, vec4 v)
{
	return rotate(r, v);
}

__device__ static vec4 world_vec_to_body(rotor4 r, vec4 v)
{
	return rotate(r.reverse(), v);
}

__device__ static vec4 body_pos_to_world(rotor4 r, vec4 position4d, vec4 v)
{
	return rotate(r, v) + position4d;
}

__device__ static vec4 world_pos_to_body(rotor4 r, vec4 position4d, vec4 v)
{
	return rotate(r.reverse(), v - position4d);
}

__device__ static vec4 body_vec_to_world(_d_RigidBody4D a, vec4 v)
{
	return rotate(a.rotation4D, v);
}

__device__ static vec4 world_vec_to_body(_d_RigidBody4D a, vec4 v)
{
	return rotate(a.rotation4D.reverse(), v);
}

__device__ static vec4 body_pos_to_world(_d_RigidBody4D a, vec4 v)
{
	return rotate(a.rotation4D, v) + a.position4d;
}

__device__ static vec4 world_pos_to_body(_d_RigidBody4D a, vec4 v)
{
	return rotate(a.rotation4D.reverse(), v - a.position4d);
}

__device__ static vec4 inverse_moment_of_mass(float mass, vec4 force4D, const float dt)
{
	return mass > 0.f ? force4D * dt / mass : vec4(0.f);
}

__device__ static bivec4 inverse_moment_of_inertia(_d_RigidBody4D a, bivec4 bodyBivec)
{
	return a.momentInertiaScalar > 0.f ? bodyBivec / a.momentInertiaScalar : bivec4();
}

__device__ static bivec4 inverse_moment_of_inertia(float momentInertiaScalar, bivec4 bodyBivec)
{
	return momentInertiaScalar > 0.f ? bodyBivec / momentInertiaScalar : bivec4();
}

__device__ static float get_moment_inertia_scalar(_d_RigidBody4D a, bivec4 angularVelocity4D)
{
	if (a.momentInertiaScalar > 0.f)
	{
		float lengthOfAngularVelocity4D(length(angularVelocity4D));
		if (lengthOfAngularVelocity4D == 0.f) {
			float radius(_d_min(a.scale4D.w, _d_min(a.scale4D.z, _d_min(a.scale4D.x, a.scale4D.y))) / 2.f);
			return a.mass * radius * radius / 5.f;
		}
		if(a.colType==3)//SPHERE4D:
		{
			float radius(_d_min(a.scale4D.w, _d_min(a.scale4D.z, _d_min(a.scale4D.x, a.scale4D.y))) / 2.f);
			return 1.5f * a.mass * radius * radius / 5.f;
		}
		else if (a.colType == 1)//BOX4D:
		{
			//std::cout << "alg::bivec4(" << angularVelocity4D.xy << "," << angularVelocity4D.xz << "," << angularVelocity4D.xw << "," << angularVelocity4D.yz << "," << angularVelocity4D.yw << "," << angularVelocity4D.zw << ")" << '\n';
			float momentInertiaScalarXY((a.scale4D.x * a.scale4D.x + a.scale4D.y * a.scale4D.y) * a.mass / 12.f);
			float momentInertiaScalarXZ((a.scale4D.x * a.scale4D.x + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarXW((a.scale4D.x * a.scale4D.x + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarYZ((a.scale4D.y * a.scale4D.y + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarYW((a.scale4D.y * a.scale4D.y + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarZW((a.scale4D.z * a.scale4D.z + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalar0(lengthOfAngularVelocity4D > 0 ? (
				_d_abs(angularVelocity4D.xy) * momentInertiaScalarXY +
				_d_abs(angularVelocity4D.xz) * momentInertiaScalarXZ +
				_d_abs(angularVelocity4D.xw) * momentInertiaScalarXW +
				_d_abs(angularVelocity4D.yz) * momentInertiaScalarYZ +
				_d_abs(angularVelocity4D.yw) * momentInertiaScalarYW +
				_d_abs(angularVelocity4D.zw) * momentInertiaScalarZW
				) / (_d_abs(angularVelocity4D.xy) + _d_abs(angularVelocity4D.xz) + _d_abs(angularVelocity4D.xw) + _d_abs(angularVelocity4D.yz) + _d_abs(angularVelocity4D.yw) + _d_abs(angularVelocity4D.zw)) : a.mass / 6.f);
			return momentInertiaScalar0;
		}
		else if (a.colType == 2)// STEP4D:
		{
			float momentInertiaScalarXY((a.scale4D.x * a.scale4D.x + a.scale4D.y * a.scale4D.y) * a.mass / 12.f);
			float momentInertiaScalarXZ((a.scale4D.x * a.scale4D.x + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarXW((a.scale4D.x * a.scale4D.x + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarYZ((a.scale4D.y * a.scale4D.y + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarYW((a.scale4D.y * a.scale4D.y + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarZW((a.scale4D.z * a.scale4D.z + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalar0(lengthOfAngularVelocity4D > 0 ? (
				_d_abs(angularVelocity4D.xy) * momentInertiaScalarXY +
				_d_abs(angularVelocity4D.xz) * momentInertiaScalarXZ +
				_d_abs(angularVelocity4D.xw) * momentInertiaScalarXW +
				_d_abs(angularVelocity4D.yz) * momentInertiaScalarYZ +
				_d_abs(angularVelocity4D.yw) * momentInertiaScalarYW +
				_d_abs(angularVelocity4D.zw) * momentInertiaScalarZW
				) / (_d_abs(angularVelocity4D.xy) + _d_abs(angularVelocity4D.xz) + _d_abs(angularVelocity4D.xw) + _d_abs(angularVelocity4D.yz) + _d_abs(angularVelocity4D.yw) + _d_abs(angularVelocity4D.zw)) : a.mass / 6.f);
			return momentInertiaScalar0;
		}
		else if (a.colType == 4)//MESH4D:
		{
			float momentInertiaScalarXY((a.scale4D.x * a.scale4D.x + a.scale4D.y * a.scale4D.y) * a.mass / 12.f);
			float momentInertiaScalarXZ((a.scale4D.x * a.scale4D.x + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarXW((a.scale4D.x * a.scale4D.x + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarYZ((a.scale4D.y * a.scale4D.y + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarYW((a.scale4D.y * a.scale4D.y + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarZW((a.scale4D.z * a.scale4D.z + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalar0(lengthOfAngularVelocity4D > 0 ? (
				_d_abs(angularVelocity4D.xy) * momentInertiaScalarXY +
				_d_abs(angularVelocity4D.xz) * momentInertiaScalarXZ +
				_d_abs(angularVelocity4D.xw) * momentInertiaScalarXW +
				_d_abs(angularVelocity4D.yz) * momentInertiaScalarYZ +
				_d_abs(angularVelocity4D.yw) * momentInertiaScalarYW +
				_d_abs(angularVelocity4D.zw) * momentInertiaScalarZW
				) / (_d_abs(angularVelocity4D.xy) + _d_abs(angularVelocity4D.xz) + _d_abs(angularVelocity4D.xw) + _d_abs(angularVelocity4D.yz) + _d_abs(angularVelocity4D.yw) + _d_abs(angularVelocity4D.zw)) : a.mass / 6.f);
			return (momentInertiaScalar0) / 1.5f;
		}
		else if (a.colType == 5)//CAPSULE4D:
		{
			float momentInertiaScalarXY((a.scale4D.x * a.scale4D.x + a.scale4D.y * a.scale4D.y) * a.mass / 12.f);
			float momentInertiaScalarXZ((a.scale4D.x * a.scale4D.x + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarXW((a.scale4D.x * a.scale4D.x + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarYZ((a.scale4D.y * a.scale4D.y + a.scale4D.z * a.scale4D.z) * a.mass / 12.f);
			float momentInertiaScalarYW((a.scale4D.y * a.scale4D.y + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalarZW((a.scale4D.z * a.scale4D.z + a.scale4D.w * a.scale4D.w) * a.mass / 12.f);
			float momentInertiaScalar0(lengthOfAngularVelocity4D > 0 ? (
				_d_abs(angularVelocity4D.xy) * momentInertiaScalarXY +
				_d_abs(angularVelocity4D.xz) * momentInertiaScalarXZ +
				_d_abs(angularVelocity4D.xw) * momentInertiaScalarXW +
				_d_abs(angularVelocity4D.yz) * momentInertiaScalarYZ +
				_d_abs(angularVelocity4D.yw) * momentInertiaScalarYW +
				_d_abs(angularVelocity4D.zw) * momentInertiaScalarZW
				) / (_d_abs(angularVelocity4D.xy) + _d_abs(angularVelocity4D.xz) + _d_abs(angularVelocity4D.xw) + _d_abs(angularVelocity4D.yz) + _d_abs(angularVelocity4D.yw) + _d_abs(angularVelocity4D.zw)) : a.mass / 6.f);
			return momentInertiaScalar0;
		}
	}
	else
	{
		return a.momentInertiaScalar;
	}
}

__device__ static void update_rigidbody_velocity4D(_d_RigidBody4D* a, float* forceSystems4D, const float dt)
{
	if (a->sizeOfForces > 0)
	{
		a->isSleep = a->isContinueSleep = false;
		for (int i(0); i < a->sizeOfForces; i++)
		{
			int forceIndex(8 * (a->forcesStartIndex + i));
			vec4 force4D = vec4(forceSystems4D[forceIndex], forceSystems4D[forceIndex + 1], forceSystems4D[forceIndex + 2], forceSystems4D[forceIndex + 3]);
			vec4 position4d = vec4(forceSystems4D[forceIndex + 4], forceSystems4D[forceIndex + 5], forceSystems4D[forceIndex + 6], forceSystems4D[forceIndex + 7]);
			a->velocity4d = a->velocity4d + (a->mass > 0.f ? force4D * dt / a->mass : vec4(0.f));
			if (position4d != vec4(0.f))
			{
				bivec4 angularVelocityAdd4D(wedge_v(position4d - a->massCenter, rotate(a->rotation4D.reverse(), force4D * dt)));
				float momentInertiaScalar(get_moment_inertia_scalar(*a, angularVelocityAdd4D));
				a->angularVelocity4D = a->angularVelocity4D + inverse_moment_of_inertia(momentInertiaScalar, angularVelocityAdd4D);
			}
		}
	}
}

//static alg::bivec4 update_angular_velocity4D(RigidBody4D* b, vec4 force4D,vec4 forcePosition4D,const float dt)
//{
//    alg::bivec4 bodyBivec(wedge_v(alg::vec4(forcePosition4D), rotate(b->rotation4D.reverse(), alg::vec4(dt * force4D))));
//    float momentInertiaScalar(get_moment_inertia_scalar(b));
//    if (b->momentInertiaScalar > 0.f) return  b->angularVelocity4D+bodyBivec / momentInertiaScalar;
//    else return b->angularVelocity4D;
//}

__device__ static void resolve_impulse(_d_RigidBody4D* b, vec4 impulse, vec4 bodyContact)
{
	b->velocity4d = b->velocity4d + (b->mass > 0.f ? impulse / b->mass : vec4(0.f));
	b->angularVelocity4D = b->angularVelocity4D + inverse_moment_of_inertia(*b, wedge_v(bodyContact - b->massCenter, rotate(b->rotation4D.reverse(), impulse)));
}

__device__ static void resolve_impulse(_d_RigidBody4D* b, vec4 impulse, vec4 bodyContact, float momentInertiaScalar)
{
	b->velocity4d = b->velocity4d + (b->mass > 0.f ? impulse / b->mass : vec4(0.f));
	b->angularVelocity4D = b->angularVelocity4D + inverse_moment_of_inertia(momentInertiaScalar, wedge_v(bodyContact - b->massCenter, rotate(b->rotation4D.reverse(), impulse)));
}

__device__ static void resolve_impulse(_d_RigidBody4D b, int index, float* velocitys4D, float* angularVelocitys4D, vec4 impulse, vec4 bodyContact, float momentInertiaScalar)
{
	vec4 velocity4DOfPlus(b.mass > 0.f ? impulse / b.mass : vec4(0.f));
	bivec4 angularVelocity4OfPlus(inverse_moment_of_inertia(momentInertiaScalar, wedge_v(bodyContact - b.massCenter, rotate(b.rotation4D.reverse(), impulse))));
	//velocitys4D[4 * index] += velocity4DOfPlus.x, velocitys4D[4 * index + 1] += velocity4DOfPlus.y, velocitys4D[4 * index + 2] += velocity4DOfPlus.z, velocitys4D[4 * index + 3] += velocity4DOfPlus.w;
	atomicAdd(&velocitys4D[4 * index], velocity4DOfPlus.x), atomicAdd(&velocitys4D[4 * index + 1], velocity4DOfPlus.y), atomicAdd(&velocitys4D[4 * index + 2], velocity4DOfPlus.z), atomicAdd(&velocitys4D[4 * index + 3], velocity4DOfPlus.w);
	//b->velocity4d = b->velocity4d + velocity4DOfPlus;
	atomicAdd(&angularVelocitys4D[6 * index], angularVelocity4OfPlus.xy), atomicAdd(&angularVelocitys4D[6 * index + 1], angularVelocity4OfPlus.xz), atomicAdd(&angularVelocitys4D[6 * index + 2], angularVelocity4OfPlus.xw), atomicAdd(&angularVelocitys4D[6 * index + 3], angularVelocity4OfPlus.yz), atomicAdd(&angularVelocitys4D[6 * index + 4], angularVelocity4OfPlus.yw), atomicAdd(&angularVelocitys4D[6 * index + 5], angularVelocity4OfPlus.zw);
	//b->angularVelocity4D = b->angularVelocity4D + angularVelocity4OfPlus;
}

__device__ static void resolve_impulse(_d_RigidBody4D* b, vec4 impulse)
{
	b->velocity4d = b->velocity4d + (b->mass > 0.f ? impulse / b->mass : vec4(0.f));
}

__device__ static void resolve_impulse0(_d_RigidBody4D* b, vec4 impulse, vec4 worldContact)
{
	b->velocity4d = b->velocity4d + (b->mass > 0.f ? impulse / b->mass : vec4(0.f));
	b->angularVelocity4D = b->angularVelocity4D + inverse_moment_of_inertia(*b, wedge_v(world_pos_to_body(*b, worldContact - b->massCenter), rotate(b->rotation4D.reverse(), impulse)));
}

__device__ static vec4 vel_at(_d_RigidBody4D b, vec4 bodyPos)
{
	vec4 rotVel(body_vec_to_world(b, left_contract_bv(bodyPos - b.massCenter, b.angularVelocity4D)));
	return b.velocity4d + rotVel;
}

__device__ static vec4 vel_at(_d_RigidBody4D b, int index, float* velocitys4D, float* angularVelocitys4D, vec4 bodyPos)
{
	vec4 velocity4d = vec4(velocitys4D[4 * index], velocitys4D[4 * index + 1], velocitys4D[4 * index + 2], velocitys4D[4 * index + 3]);
	bivec4 angularVelocity4D = bivec4(angularVelocitys4D[6 * index], angularVelocitys4D[6 * index + 1], angularVelocitys4D[6 * index + 2], angularVelocitys4D[6 * index + 3], angularVelocitys4D[6 * index + 4], angularVelocitys4D[6 * index + 5]);
	vec4 rotVel(body_vec_to_world(b, left_contract_bv(bodyPos - b.massCenter, angularVelocity4D)));
	return velocity4d + rotVel;
}

__device__ static vec4 vel_at_world(_d_RigidBody4D b, vec4 worldPos)
{
	vec4 bodyPos(world_pos_to_body(b, worldPos));
	vec4 rotVel(body_vec_to_world(b, left_contract_bv(bodyPos - b.massCenter, b.angularVelocity4D)));
	return b.velocity4d + rotVel;
}

__device__ static struct _d_Edge4D
{
	vec4 vertices4D[2];
};

__device__ static struct _d_Face4D
{
	vec4 vertices4D[3];
};

__device__ static struct _d_Tetra4D
{
	vec4 vertices4D[4];
};

__device__ static struct _d_Cube4D
{
	vec4 vertices4D[4];
	vec4 normal4D;
	int direction4D;
};

__device__ static vec4 hypercube_vertices4D(int n)
{
	if (n == 0)return vec4(-0.5f, -0.5f, 0.5f, -0.5f);
	else if (n == 1)return vec4(0.5f, -0.5f, 0.5f, -0.5f);
	else if (n == 2)return vec4(0.5f, -0.5f, -0.5f, -0.5f);
	else if (n == 3)return vec4(-0.5f, -0.5f, -0.5f, -0.5f);
	else if (n == 4)return vec4(-0.5f, 0.5f, 0.5f, -0.5f);
	else if (n == 5)return vec4(0.5f, 0.5f, 0.5f, -0.5f);
	else if (n == 6)return vec4(0.5f, 0.5f, -0.5f, -0.5f);
	else if (n == 7)return vec4(-0.5f, 0.5f, -0.5f, -0.5f);
	else if (n == 8)return vec4(-0.5f, -0.5f, 0.5f, 0.5f);
	else if (n == 9)return vec4(0.5f, -0.5f, 0.5f, 0.5f);
	else if (n == 10)return vec4(0.5f, -0.5f, -0.5f, 0.5f);
	else if (n == 11)return vec4(-0.5f, -0.5f, -0.5f, 0.5f);
	else if (n == 12)return vec4(-0.5f, 0.5f, 0.5f, 0.5f);
	else if (n == 13)return vec4(0.5f, 0.5f, 0.5f, 0.5f);
	else if (n == 14)return vec4(0.5f, 0.5f, -0.5f, 0.5f);
	else if (n == 15)return vec4(-0.5f, 0.5f, -0.5f, 0.5f);
}

__device__ static _d_Edge4D edges4D(int n)
{
	if (n == 0)return { vec4(0.5f, 0.5f, 0.5f, 0.5f), vec4(-0.5f, 0.5f, 0.5f, 0.5f) };//0
	else if (n == 1)return { vec4(0.5f, -0.5f, 0.5f, 0.5f), vec4(-0.5f, -0.5f, 0.5f, 0.5f) };
	else if (n == 2)return { vec4(0.5f, 0.5f, -0.5f, 0.5f), vec4(-0.5f, 0.5f, -0.5f, 0.5f) };
	else if (n == 3)return { vec4(0.5f, 0.5f, 0.5f, -0.5f), vec4(-0.5f, 0.5f, 0.5f, -0.5f) };
	else if (n == 4)return { vec4(0.5f, -0.5f, -0.5f, 0.5f), vec4(-0.5f, -0.5f, -0.5f, 0.5f) };
	else if (n == 5)return { vec4(0.5f, -0.5f, 0.5f, -0.5f), vec4(-0.5f, -0.5f, 0.5f, -0.5f) };
	else if (n == 6)return { vec4(0.5f, 0.5f, -0.5f, -0.5f), vec4(-0.5f, 0.5f, -0.5f, -0.5f) };
	else if (n == 7)return { vec4(0.5f, -0.5f, -0.5f, -0.5f), vec4(-0.5f, -0.5f, -0.5f, -0.5f) };

	else if (n == 8)return { vec4(0.5f, 0.5f, 0.5f, 0.5f), vec4(0.5f, -0.5f, 0.5f, 0.5f) };//8
	else if (n == 9)return { vec4(-0.5f, 0.5f, 0.5f, 0.5f), vec4(-0.5f, -0.5f, 0.5f, 0.5f) };
	else if (n == 10)return { vec4(0.5f, 0.5f, -0.5f, 0.5f), vec4(0.5f, -0.5f, -0.5f, 0.5f) };
	else if (n == 11)return { vec4(0.5f, 0.5f, 0.5f, -0.5f), vec4(0.5f, -0.5f, 0.5f, -0.5f) };
	else if (n == 12)return { vec4(-0.5f, 0.5f, -0.5f, 0.5f), vec4(-0.5f, -0.5f, -0.5f, 0.5f) };
	else if (n == 13)return { vec4(-0.5f, 0.5f, 0.5f, -0.5f), vec4(-0.5f, -0.5f, 0.5f, -0.5f) };
	else if (n == 14)return { vec4(0.5f, 0.5f, -0.5f, -0.5f), vec4(0.5f, -0.5f, -0.5f, -0.5f) };
	else if (n == 15)return { vec4(-0.5f, 0.5f, -0.5f, -0.5f), vec4(-0.5f, -0.5f, -0.5f, -0.5f) };

	else if (n == 16)return { vec4(0.5f, 0.5f,  0.5f,0.5f), vec4(0.5f, 0.5f,  -0.5f,0.5f) }; //16
	else if (n == 17)return { vec4(-0.5f, 0.5f, 0.5f,0.5f), vec4(-0.5f, 0.5f, -0.5f,0.5f) };
	else if (n == 18)return { vec4(0.5f, -0.5f, 0.5f,0.5f), vec4(0.5f, -0.5f, -0.5f,0.5f) };
	else if (n == 19)return { vec4(0.5f, 0.5f, 0.5f, -0.5f), vec4(0.5f, 0.5f, -0.5f, -0.5f) };
	else if (n == 20)return { vec4(-0.5f, -0.5f, 0.5f, 0.5f), vec4(-0.5f, -0.5f, -0.5f, 0.5f) };
	else if (n == 21)return { vec4(-0.5f, 0.5f, 0.5f,-0.5f),vec4(-0.5f, 0.5f, -0.5f,-0.5f) };
	else if (n == 22)return { vec4(0.5f, -0.5f, 0.5f,-0.5f),vec4(0.5f, -0.5f, -0.5f,-0.5f) };
	else if (n == 23)return { vec4(-0.5f, -0.5f,0.5f,-0.5f),vec4(-0.5f, -0.5f,-0.5f,-0.5f) };

	else if (n == 24)return { vec4(0.5f, 0.5f,  0.5f ,0.5f),vec4(0.5f, 0.5f,  0.5f ,-0.5f) }; //24
	else if (n == 25)return { vec4(-0.5f, 0.5f, 0.5f ,0.5f),vec4(-0.5f, 0.5f, 0.5f ,-0.5f) };
	else if (n == 26)return { vec4(0.5f, -0.5f, 0.5f ,0.5f),vec4(0.5f, -0.5f, 0.5f ,-0.5f) };
	else if (n == 27)return { vec4(0.5f, 0.5f,  -0.5f,0.5f),vec4(0.5f, 0.5f,  -0.5f,-0.5f) };
	else if (n == 28)return { vec4(-0.5f, -0.5f,0.5f ,0.5f),vec4(-0.5f, -0.5f,0.5f ,-0.5f) };
	else if (n == 29)return { vec4(-0.5f, 0.5f, -0.5f,0.5f),vec4(-0.5f, 0.5f, -0.5f,-0.5f) };
	else if (n == 30)return { vec4(0.5f, -0.5f, -0.5f,0.5f),vec4(0.5f, -0.5f, -0.5f,-0.5f) };
	else if (n == 31)return { vec4(-0.5f, -0.5f,-0.5f,0.5f),vec4(-0.5f, -0.5f,-0.5f,-0.5f) };
}
__device__ static _d_Face4D faces4D(int n)
{
	if(n==0) return {vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,0.5f)};
	else if(n==1) return {vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f)};
	else if(n==2) return {vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f)};
	else if(n==3) return {vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f)};
	else if(n==4) return {vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f)};
	else if(n==5) return {vec4(0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f)};
	else if(n==6) return {vec4(0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, 0.5f, -0.5f,-0.5f)};
	else if(n==7) return {vec4(0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, 0.5f,-0.5f)};
	else if(n==8) return {vec4(0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,0.5f)};
	else if(n==9) return {vec4(-0.5f, 0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, 0.5f,-0.5f)};
	else if(n==10) return {vec4(-0.5f, 0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,0.5f)};
	else if(n==11) return {vec4(-0.5f, -0.5f, 0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,0.5f)};
	else if(n==12) return {vec4(-0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, -0.5f, -0.5f,0.5f)};
	else if(n==13) return {vec4(-0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, -0.5f,0.5f)};
	else if(n==14) return {vec4(-0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,-0.5f)};
	else if(n==15) return {vec4(-0.5f, -0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, -0.5f,0.5f)};
	else if(n==16) return {vec4(-0.5f, -0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,-0.5f)};
	else if(n==17) return {vec4(-0.5f, 0.5f, -0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,-0.5f)};
	else if(n==18) return {vec4(-0.5f, 0.5f, 0.5f,-0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f),vec4(0.5f, -0.5f, 0.5f,-0.5f)};
	else if(n==19) return {vec4(-0.5f, 0.5f, 0.5f,-0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f),vec4(0.5f, 0.5f, -0.5f,-0.5f)};
	else if(n==20) return {vec4(-0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,-0.5f)};
	else if(n==21) return {vec4(0.5f, -0.5f, 0.5f,-0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f),vec4(0.5f, 0.5f, -0.5f,-0.5f)};
	else if(n==22) return {vec4(0.5f, -0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,-0.5f)};
	else if(n==23) return {vec4(0.5f, -0.5f, -0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,-0.5f)};
}
__device__ static _d_Cube4D cubes4D(int n)
{
	if(n==0) return { vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, 0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, 0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,0.5f),vec4(-1.f, 0.f, 0.f,0.f),0};
	else if(n==1) return { vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, 0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,0.5f),vec4(0.f, -1.f, 0.f,0.f),1};
	else if(n==2) return { vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, 0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, -0.5f,0.5f),vec4(0.f, 0.f, -1.f,0.f),2};
	else if(n==3) return { vec4(-0.5f, -0.5f, -0.5f,-0.5f),vec4(0.5f, -0.5f, -0.5f,-0.5f),vec4(-0.5f, 0.5f, -0.5f,-0.5f),vec4(-0.5f, -0.5f, 0.5f,-0.5f),vec4(0.f, 0.f, 0.f,-1.f),3};
	else if(n==4) return { vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f),vec4(1.f, 0.f, 0.f,0.f),4};
	else if(n==5) return { vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f),vec4(0.f, 1.f, 0.f,0.f),5};
	else if(n==6) return { vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, 0.5f,-0.5f),vec4(0.f, 0.f, 1.f,0.f),6};
	else if(n==7) return { vec4(0.5f, 0.5f, 0.5f,0.5f),vec4(-0.5f, 0.5f, 0.5f,0.5f),vec4(0.5f, -0.5f, 0.5f,0.5f),vec4(0.5f, 0.5f, -0.5f,0.5f),vec4(0.f, 0.f, 0.f,1.f),7};
}

__device__ static vec4 get_vertex(int primitiveId, int index, float* vertices4DsVec, int* verticesStartIndices)
{
	index = 4 * (verticesStartIndices[primitiveId] + index);
	return vec4(vertices4DsVec[index], vertices4DsVec[index + 1], vertices4DsVec[index + 2], vertices4DsVec[index + 3]);
}